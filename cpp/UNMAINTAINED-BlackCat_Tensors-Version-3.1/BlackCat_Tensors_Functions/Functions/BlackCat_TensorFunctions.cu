#ifndef BLACKCAT_TENSOR_FUNCTIONS
#define BLACKCAT_TENSOR_FUNCTIONS
namespace BC {

#ifdef __HIPCC__
#define __BC_gcpu__ __host__ __device__
#define BLACKCAT_GPU_ENABLED
#else
#define __BC_gcpu__
#endif

	namespace NN_Functions {

		struct Sigmoid {

			template<class T>
			__BC_gcpu__ inline __attribute__((always_inline))  T operator () (T t) const {
				static constexpr T e = 2.71828;

				return 1 / (1 + std::pow(e, - t));
			}
		};
		struct SigmoidAssign {

			template<class T>
			__BC_gcpu__ inline __attribute__((always_inline))  T operator () (T& t) const {
				static constexpr T e = 2.71828;

				return t = 1 / (1 + std::pow(e, - t));
			}
		};
		struct CachedSigmoidDeriv {

			template<class T>
			__BC_gcpu__ inline __attribute__((always_inline))  T operator () (T t) const {
				return t * (1 - t);
			}
		};
		struct CachedSigmoidDerivAssign {

			template<class T>
			__BC_gcpu__ inline __attribute__((always_inline))  T operator () (T& t) const {
				return t *= (1 - t);
			}
		};

		struct Tanh {

			template<class T>
			__BC_gcpu__ inline __attribute__((always_inline))  T operator () (T t) const {
				static constexpr double e = 2.71828;

				return (powf(e, t) - powf(e, -t)) /
					   (powf(e, t) + powf(e, -t));
			}
		};
		struct TanhAssign {

			template<class T>
			__BC_gcpu__ inline __attribute__((always_inline))  T operator () (T& t) const {
				static constexpr T e = 2.71828;

				return t = (powf(e, t) - powf(e, -t)) /
					       (powf(e, t) + powf(e, -t));
			}
		};
		struct CachedTanhDeriv {

			template<class T>
			__BC_gcpu__ inline __attribute__((always_inline))  T operator () (T t) const {
				return 1 - powf(t, 2);
			}
		};
		struct CachedTanhDerivAssign {

			template<class T>

			__BC_gcpu__ inline __attribute__((always_inline))  T operator () (T& t) const {
				static constexpr T e = 2.71828;

				return t = 1 - powf(t, 2);
			}
		};

		template<template<class,class> class tensor, class T, class ml>
		auto sigmoid(tensor<T, ml>& x) {
			return x.unExpr(SigmoidAssign());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto sigmoid(tensor<T, ml>&& x) {
			return x.unExpr(Sigmoid());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto sigmoidDeriv(tensor<T, ml>& x) {
			return x.unExpr(CachedSigmoidDerivAssign());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto sigmoidDeriv(tensor<T, ml>&& x) {
			return x.unExpr(CachedSigmoidDeriv());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto tanh(tensor<T, ml>& x) {
			return x.unExpr(TanhAssign());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto tanh(tensor<T, ml>&& x) {
			return x.unExpr(Tanh());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto tanhDeriv(tensor<T, ml>& x) {
			return x.unExpr(CachedTanhDerivAssign());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto tanhDeriv(tensor<T, ml>&& x) {
			return x.unExpr(CachedTanhDeriv());
		}

	}

	namespace NN_Abreviated_Functions {
		using namespace NN_Functions;
		template<template<class,class> class tensor, class T, class ml>
		auto g(tensor<T, ml>& x) {
			return x.unExpr(SigmoidAssign());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto g(tensor<T, ml>&& x) {
			return x.unExpr(Sigmoid());
		}
//		template<template<class,class> class tensor, class T, class ml>
//		auto gd(tensor<T, ml>& x) {
//			return x.unExpr(CachedSigmoidDerivAssign());
//		}
		template<template<class,class> class tensor, class T, class ml>
		auto gd(tensor<T, ml>& x) {
			return x.unExpr(CachedSigmoidDeriv());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto h(tensor<T, ml>& x) {
			return x.unExpr(TanhAssign());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto h(tensor<T, ml>&& x) {
			return x.unExpr(Tanh());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto hd(tensor<T, ml>& x) {
			return x.unExpr(CachedTanhDerivAssign());
		}
		template<template<class,class> class tensor, class T, class ml>
		auto hd(tensor<T, ml>&& x) {
			return x.unExpr(CachedTanhDeriv());
		}

	}

}
#endif
