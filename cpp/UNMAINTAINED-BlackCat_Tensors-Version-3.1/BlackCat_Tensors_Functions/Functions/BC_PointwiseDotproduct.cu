/*
 * BC_PointwiseDotproduct.cu
 *
 *  Created on: Feb 6, 2018
 *      Author: joseph
 */

#ifndef BC_POINTWISEDOTPRODUCT_CU_
#define BC_POINTWISEDOTPRODUCT_CU_

namespace BC {
#ifdef __HIPCC__
#define __BC_gcpu__ __host__ __device__
#define BLACKCAT_GPU_ENABLED
#else
#define __BC_gcpu__
#endif

template<class, class> class Matrix;
template<class, class> class Vector;
template<class, class> class expression;

template<class T, class functor_type>
struct pointwise_accessor : expression<T, pointwise_accessor<T, functor_type>> {
	//Evaluates a single row*column operation of two tensors

	functor_type lv;
	functor_type rv;

	T*	evaluation;
};


template<class T, class ml>
Vector<T, ml> operator * (const Matrix<T, ml>&, const Vector<T, ml>&) {

}






}


#endif /* BC_POINTWISEDOTPRODUCT_CU_ */
