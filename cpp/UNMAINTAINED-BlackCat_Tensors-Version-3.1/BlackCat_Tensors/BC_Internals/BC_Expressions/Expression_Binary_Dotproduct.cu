/*
 * Expression_Binary_Dotproduct.cu
 *
 *  Created on: Jan 9, 2018
 *      Author: joseph
 */
#ifdef  __HIPCC__
#ifndef EXPRESSION_BINARY_DOTPRODUCT_CU_
#define EXPRESSION_BINARY_DOTPRODUCT_CU_


#include "Expression_Binary_Dotproduct_impl.cu"
#include "Expression_Base.cu"
#include "../BlackCat_Internal_Definitions.h"
#include "../BC_MetaTemplateFunctions/Adhoc.h"
#include "../BC_Core/Implementation_Core/Shape.h"
#include <iostream>
#include <memory>

namespace BC {

/*
 * a = M x K
 * b = K x N
 * c = M x N
 */

//The evaluation of dot_products are tightly coupled with the unary expression - transpose, and binary expression scalar L and scalar R.T*
//This is just because CUDA doesn't have constexpr_if support yet. Get on it guys.
template<class T, class U>
void assignDP(T* from, U to) {
	throw std::invalid_argument("Illegal Assign");
}
template<class T, class lv, class rv>
void assignDP(T*& from, binary_expression<T, assign, lv, rv>& bed) {
	from = bed.left;
}
template<class T>
void assignDP(T*& from, T* to) {
	from = to;
}


template<class T, class lv, class rv, class Mathlibrary>
struct binary_expression_dotproduct : non_linear_expression<T, binary_expression_dotproduct<T, lv, rv, Mathlibrary>> {
	using scalar_type = typename MTF::determine_scalar<T>::type;

	//this is the deleter for the shared ptr --> We use a shared pointer so that everytime you pass the extended function of a dot product
	//the program does not recalculate the dot_product //Normally I hate shared pointers.
	struct deleter {
		void operator()(const scalar_type* t) {
			Mathlibrary::destroy(t);
		}
	};

	static constexpr int ALPHA_DEFAULT  = 1;
	static constexpr int BETA_DEFAULT   = 0;

	const lv& left;
	const rv& right;

	const int M = 0;
	const int N = 0;
	const int K = 0;
	const int LDA = 0;
	const int LDB = 0;
	const int LDC = 0;

	static constexpr bool lv_needs_to_be_evaluated = evaluate<lv>::conditional;
	static constexpr bool rv_needs_to_be_evaluated = evaluate<rv>::conditional;
	static constexpr bool transA = evaluate<lv>::transpose;
	static constexpr bool transB = evaluate<rv>::transpose;

	using l_eval_array = typename MTF::determine_scalar<lv>::type;
	using r_eval_array = typename MTF::determine_scalar<rv>::type;

	const int lv_size = M * K;
	const int rv_size = N * K;
	const int eval_size = M * N ;

	bool parent = true;
	std::shared_ptr<scalar_type> array;
	scalar_type* array_ptr = array.get();

	__attribute__((always_inline))
	binary_expression_dotproduct(const lv& left, const rv& right, int M, int N, int K, int ldM = 0, int ldN = 0, int ldK = 0) :
	M(M), N(N), K(K),
	LDA(ldM == 0 ? M : ldM), LDB(ldN == 0 ? K : ldN), LDC(ldK == 0 ? M : ldK),
	left(left), right(right) {

		eval();
		array_ptr = array.get();
	}


public:

	void eval() {
		auto A_unevaluated = evaluate<lv>::getArray(left);
		auto B_unevaluated = evaluate<rv>::getArray(right);
		scalar_type* scal_A = evaluate<lv>::getScalar(left);
		scalar_type* scal_B = evaluate<rv>::getScalar(right);

		T* A;
		T* B;

		if (lv_needs_to_be_evaluated) {
			if (self_eval<lv>::conditional) {
				Mathlibrary::eval(A_unevaluated, lv_size);
				assignDP(A, A_unevaluated);
			} else {
			Mathlibrary::initialize(A, lv_size);
			Mathlibrary::copy(A, A_unevaluated, lv_size);
			}
		}
			else { assignDP(A, A_unevaluated); }

		if (rv_needs_to_be_evaluated) {
			if (self_eval<rv>::conditional) {
				Mathlibrary::eval(B_unevaluated, rv_size);
				assignDP(B, B_unevaluated);
			} else {
				Mathlibrary::initialize(B, rv_size);
				Mathlibrary::copy(B, B_unevaluated, rv_size);
			}
		}
			else { assignDP(B, B_unevaluated); }


		scalar_type* tmp;
		Mathlibrary::initialize(tmp, eval_size);
		array = std::shared_ptr<scalar_type>(tmp, deleter());

		//If both sides are multiplied by a scalar --- multiply the scalars
		if (scal_A && scal_B) {
			T* scalar;
			Mathlibrary::initialize(scalar, 1);
			Mathlibrary::scalarMul(scalar, scal_A, scal_B);
			Mathlibrary::MatrixMul(transA, transB, A, B, array.get(), M, N, K, scalar, nullptr, LDA, LDB, LDC);
			Mathlibrary::destroy(scalar);
		} else if (scal_A) {
		Mathlibrary::MatrixMul(transA, transB, A, B, array.get(), M, N, K, scal_A, nullptr, LDA, LDB, LDC);
		} else if (scal_B){
			scal_A = scal_B;
			scal_B = nullptr;
			Mathlibrary::MatrixMul(transA, transB, A, B, array.get(), M, N, K, scal_A, nullptr, LDA, LDB, LDC);
		} else {
			Mathlibrary::MatrixMul(transA, transB, A, B, array.get(), M, N, K, nullptr, nullptr, LDA, LDB, LDC);
		}

//				std::cout << "dotproduct stats --------------------------------------------------------------------------" << std::endl;
//						std::cout << " m n k = " << M << "  "<< N << " " << K << std::endl;
//
//						if (lv_needs_to_be_evaluated) {
//							if (self_eval<lv>::conditional) {
//								std::cout << " lv self eval " << std::endl;
//							} else
//							std::cout << " lv was evaluated " << std::endl;
//						}
//						if (rv_needs_to_be_evaluated) {
//							if (self_eval<rv>::conditional) {
//								std::cout << " rv self eval " << std::endl;
//							} else
//							std::cout << " rv was evaluated " << std::endl;
//						}
//						if (transA) {
//							std::cout << " A - fast trans " << std::endl;
//						}
//						if (transB) {
//							std::cout << " B - fast trans " << std::endl;
//						}
//						if (evaluate<lv>::scalar) {
//							std::cout << " lv scalar detected " << std::endl;
//						}
//						if (evaluate<rv>::scalar) {
//							std::cout << " rv scalar detected " << std::endl;
//						}
//						if (scal_A && scal_B)
//						std::cout << "scalars = " << *scal_A <<  " " << *scal_B << std::endl;
//						std::cout << " --------------------------------------------------------------------------" << std::endl;

		if (lv_needs_to_be_evaluated && !self_eval<lv>::conditional) { Mathlibrary::destroy(A); }
		if (rv_needs_to_be_evaluated && !self_eval<rv>::conditional) { Mathlibrary::destroy(B); }

	}

		__attribute__((always_inline))  __BC_gcpu__
		 const T operator [](int index) const {
			return array_ptr[index];
		}
		__attribute__((always_inline))  __BC_gcpu__
			T operator [](int index) {
			return array_ptr[index];
		}
	};
}

#endif /* EXPRESSION_BINARY_DOTPRODUCT_CU_ */
#endif


//				Some printouts for debugging
//
//		std::cout << "dotproduct stats --------------------------------------------------------------------------" << std::endl;
//				std::cout << " m n k = " << M << "  "<< N << " " << K << std::endl;
//
//				if (lv_needs_to_be_evaluated) {
//					if (self_eval<lv>::conditioanl) {
//						std::cout << " lv self eval " << std::endl;
//					}
//					std::cout << " lv was evaluated " << std::endl;
//				}
//				if (rv_needs_to_be_evaluated) {
//					if (self_eval<rv>::conditioanl) {
//						std::cout << " rv self eval " << std::endl;
//					}
//					std::cout << " lv was evaluated " << std::endl;
//				}
//				if (transA) {
//					std::cout << " A - fast trans " << std::endl;
//				}
//				if (transB) {
//					std::cout << " B - fast trans " << std::endl;
//				}
//				if (evaluate<lv>::scalar) {
//					std::cout << " lv scalar detected " << std::endl;
//				}
//				if (evaluate<rv>::scalar) {
//					std::cout << " rv scalar detected " << std::endl;
//				}
//				if (scal_A && scal_B)
//				std::cout << "scalars = " << *scal_A <<  " " << *scal_B << std::endl;
//				std::cout << " --------------------------------------------------------------------------" << std::endl;


