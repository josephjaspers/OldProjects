#ifdef __HIPCC__
#ifndef MATHEMATICS_GPU_H_
#define MATHEMATICS_GPU_H_
#include "Mathematics_GPU_impl.cu"

#include <cmath>
#include <iostream>
#include <string>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "BC_PrintFunctions.h"

#include "../BlackCat_Internal_Definitions.h"
#include "BC_PrintFunctions.h"
#include <hipblas.h>

#include "Mathematics_GPU_impl.cu"

namespace BC {

namespace Constants {

static float* static_initialize(int sz, float value) {
	float* t;
	hipMallocManaged((void**) &t, sizeof(float));
	hipMemcpy(t, &value, sizeof(float), hipMemcpyHostToDevice);
	return t;
}

static const float* const BC_ONE = static_initialize(1, 1);
static const float* const BC_ZERO = static_initialize(1, 1);

}

class GPU {
public:

	static int blocks(int size) {
		return (size + CUDA_BASE_THREADS - 1) / CUDA_BASE_THREADS;
	}
	static int threads() {
		return CUDA_BASE_THREADS;
	}

	template<typename T>
	static void initialize(T*& t, int sz) {
		hipMalloc((void**) &t, sizeof(T) * sz);
	}

	template<class T>
	static void HostToDevice(T* t, const T* u, int size) {
		hipMemcpy(t, u, sizeof(T) * size, hipMemcpyHostToDevice);
	}
	template<class T>
	static void DeviceToHost(T* t, const T* u, int size) {
		hipMemcpy(t, u, sizeof(T) * size, hipMemcpyDeviceToHost);
	}

	template<typename T>
	static void unified_initialize(T*& t, int sz) {
		hipMallocManaged((void**) &t, sizeof(T) * sz);
	}

	template<class T, class U>
	static void copy(T t, U u, int sz) {
		gpu_impl::copy<<<blocks(sz),threads()>>>(t, u, sz);
		hipDeviceSynchronize();
	}

	template<class T, template<class...> class U, class... set>
	static void copy(T t, U<set...> u, int sz) {
		gpu_impl::copy<<<blocks(sz),threads()>>>(t, u, sz);
		hipDeviceSynchronize();
	}

	template<class T>
	static void eval(T t, int sz) {
		gpu_impl::eval<<<blocks(sz),threads()>>>(t, sz);
		hipDeviceSynchronize();
	}

	template<class U, class T, class V>
	static void scalarMul(U eval, T a, V b) {
		gpu_impl::scalarMul<<<1, 1>>>(eval, a, b);
		hipDeviceSynchronize();
	}

	template<typename T>
	static void destroy(T* t) {
		hipFree((void*)t);
	}
	template<typename T>
	static void destroy(T t) {
		throw std::invalid_argument("destruction on class object");
	}
	template<typename T, typename J>
	static void fill(T& t, const J j, int sz) {
		gpu_impl::fill<<<blocks(sz),threads()>>>(t, j, sz);
		hipDeviceSynchronize();
	}

	template<typename T, typename J>
	static void fill(T& t, const J* j, int sz) {
		gpu_impl::fill<<<blocks(sz),threads()>>>(t, j, sz);
		hipDeviceSynchronize();
	}
	template<typename T>
	static void zero(T& t, int sz) {
		gpu_impl::zero<<<blocks(sz),threads()>>>(t, sz);
		hipDeviceSynchronize();
	}

	template<typename T, class J>
	static void randomize(T t, J lower_bound, J upper_bound, int sz) {
		gpu_impl::randomize<<<blocks(sz),threads()>>>(t, lower_bound, upper_bound, sz, rand());
		hipDeviceSynchronize();
	}

	// Multiply the arrays A and B on GPU and save the result in C
	// C(m,n) = A(m,k) * B(k,n)
//A ld = m
//B ld = k
//C ld = m

	static void MatrixMul(bool transA, bool transB, const float *A, const float *B, float *C, const int m, const int n, const int k, const float* scalarA = nullptr, const float* scalarB = nullptr, int lda = 0, int ldb = 0, int ldc = 0) {
		if (lda == 0 ) lda = m;
		if (ldb == 0 ) ldb = k;
		if (ldc == 0 ) ldc = m;

		hipblasHandle_t handle;
		hipblasCreate(&handle);
		hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);

		auto TRANS_A = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
		auto TRANS_B = transA ? HIPBLAS_OP_T : HIPBLAS_OP_N;

		const float *const alpha = scalarA ? const_cast<float*>(scalarA) : Constants::BC_ONE;  //assign the scalar
		const float *const beta = scalarB ? const_cast<float*>(scalarB) : Constants::BC_ZERO;//same

		hipblasSgemm(handle, TRANS_A, TRANS_B, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
		hipDeviceSynchronize();
		hipblasDestroy(handle);
	}

	template<class ranks>
	static int calc_size(ranks R, int order) {
		if (order == 0) {
			return 1;
		}

		int sz = 1;
		for (int i = 0; i < order; ++i) {
			sz *= R[i];
		}
		return sz;
	}

	template<class T, class RANKS>
	static void print(const T* ary, const RANKS ranks, int order, int print_length) {
		int sz = calc_size(ranks, order);
		T* print = new T[sz];

		hipMemcpy(print, ary, sizeof(T) * sz, hipMemcpyDeviceToHost);

		BC::print(print, ranks, order, print_length);
		delete[] print;
	}
	template<class T, class RANKS>
	static void printSparse(const T* ary, const RANKS ranks, int order, int print_length) {
		int sz = calc_size(ranks, order);
		T* print = new T[sz];

		hipMemcpy(print, ary, sizeof(T) * sz, hipMemcpyDeviceToHost);

		BC::print(print, ranks, order, print_length);
		delete[] print;
	}
};

}

#endif /* MATHEMATICS_CPU_H_ */

#endif //if cudda cc defined
