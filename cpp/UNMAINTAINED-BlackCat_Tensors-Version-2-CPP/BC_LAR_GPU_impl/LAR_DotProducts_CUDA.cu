
#include <hip/hip_runtime.h>
//#include "BLACKCAT_GPU_MATHEMATICS.cuh"
//
//__global__
//void GPU_MATHEMATICS::dot(float* store, unsigned s_LD, const float* m1, unsigned m1_r, unsigned m1_c, unsigned m1_LD,
//																			 const float* m2, unsigned m2_r, unsigned m2_c, unsigned m2_LD)
//{
////	float* scal_one;
////	cudaMalloc(&scal_one, sizeof(float));
////
////    cublasHandle_t h;
////    cublasCreate(&h);
////    cublasSetPointerMode(h, CUBLAS_POINTER_MODE_DEVICE);
////	cublasSgemm(h,
////			CUBLAS_OP_N,
////			CUBLAS_OP_N,
////            m1_r, //height
////            m2_c, //width
////            m1_c,//wdith
////            scal_one, //alpha
////            m1, //d matb
////            m1_LD, //matb ld
////
////            m2, //d mat a
////            m2_LD, //mat a ld
////            scal_one, //beta scal
////            store, //dmat x
////            s_LD); //ld
////
////	cudaFree(scal_one);
//}
//
