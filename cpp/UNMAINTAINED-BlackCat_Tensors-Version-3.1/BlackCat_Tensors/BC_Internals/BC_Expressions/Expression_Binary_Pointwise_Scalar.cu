
/*
 * BC_Expression_Binary_Pointwise_ScalarL.h
 *
 *  Created on: Dec 2, 2017
 *      Author: joseph
 */
#ifdef  __HIPCC__
#ifndef EXPRESSION_BINARY_POINTWISE_SCALAR_H_
#define EXPRESSION_BINARY_POINTWISE_SCALAR_H_

#include "Expression_Base.cu"
#include "../BlackCat_Internal_Definitions.h"

namespace BC {

template<class T, class operation, class lv, class rv>
class binary_expression_scalar_L : expression<T,binary_expression_scalar_L<T, operation, lv, rv>> {
public:

	using this_type = binary_expression_scalar_L<T, operation, lv, rv>;

	operation oper;

	lv left;
	rv right;

	inline __attribute__((always_inline))  __BC_gcpu__ binary_expression_scalar_L(lv l, rv r) : left(l), right(r) {}
	inline __attribute__((always_inline))  __BC_gcpu__ auto operator [](int index) const { return oper(left[0], right[index]); }
};

template<class T, class operation, class lv, class rv>
class binary_expression_scalar_R : expression<T, binary_expression_scalar_R<T, operation, lv, rv>> {
public:

	using this_type = binary_expression_scalar_R<T, operation, lv, rv>;

	operation oper;

	lv left;
	rv right;

	inline __attribute__((always_inline))  __BC_gcpu__ binary_expression_scalar_R(lv l, rv r) : left(l), right(r) {}
	inline __attribute__((always_inline))  __BC_gcpu__ auto operator [](int index) const { return oper(left[index], right[0]);}
};

template<class T, class operation, class lv, class rv>
class binary_expression_scalar_LR : expression<T, binary_expression_scalar_LR<T, operation, lv, rv>> {
public:
			lv left;
			rv right;
			operation oper;


			template<class ml>
			inline __attribute__((always_inline))  __BC_gcpu__ binary_expression_scalar_LR(lv l, rv r, ml lib) : left(l), right(r) {}
			inline __attribute__((always_inline))  __BC_gcpu__ auto operator [](int index) const { return oper(left[0], right[0]); }
};
}

#endif /* EXPRESSION_BINARY_POINTWISE_SCALAR_H_ */
#endif
