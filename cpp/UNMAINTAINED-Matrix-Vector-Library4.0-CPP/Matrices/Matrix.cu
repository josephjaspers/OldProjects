#include "hip/hip_runtime.h"
#include "Matrix.h"


void Matrix::initialize()
{
#define CHK_ERROR if (erro != hipSuccess) goto Error;

	auto erro = hipMalloc(&gpu_matrix, sizeof(double) * sz);
	cpu_matrix = new double[sz];


	if (erro)
		Error: std::cerr << "Error on CUDA: " << hipGetErrorString(erro);
}

bool Matrix::updateCPU() const
{
	//hipMemcpy(device, host, bytes, hipMemcpyHostToDevice);
	//hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost);
	
	if (!gpu_update) {
		return false;
	}

	auto erro = hipMemcpy(cpu_matrix, gpu_matrix, sizeof(double) * sz, hipMemcpyDeviceToHost);
	if (erro)
		Error: std::cerr << "Error on CUDA: " << hipGetErrorString(erro);

	cpu_update = false;
	gpu_update = false;

	hipDeviceSynchronize();
	return true;
}

bool Matrix::updateGPU() const
{
	if (!cpu_update) {
		return false;
	}
	//hipMemcpy(device, host, bytes, hipMemcpyHostToDevice);
	//hipMemcpy(host, device, bytes, hipMemcpyDeviceToHost);

	auto erro =	hipMemcpy(gpu_matrix, cpu_matrix, sizeof(double) * sz, hipMemcpyHostToDevice);
	if (erro)
	Error: std::cerr << "Error on CUDA: " << hipGetErrorString(erro);

	cpu_update = false;
	gpu_update = false;

	hipDeviceSynchronize();
	return true;
}



void Matrix::assert_cpu_update(bool update)
{
	cpu_update = update;
}

void Matrix::assert_gpu_update(bool update)
{
	gpu_update = true;
}

void Matrix::force_cpu_update()
{
	gpu_update = true; 
	updateCPU();
}

void Matrix::force_gpu_update()
{
	cpu_update = true;
	updateGPU();
}

Matrix::Matrix(unsigned int length, unsigned int width)
{
	x_dim = length;
	y_dim = width;
	sz = x_dim * y_dim;

	initialize();
}

Matrix::Matrix(std::vector<double> m, int x, int y)
{
	x_dim = x;
	y_dim = y;
	sz = x * y;

	initialize();

	for (int i = 0; i < m.size(); ++i) {
		(*this)[i] = m[i];
	}
	cpu_update = true;
	updateGPU();
}

Matrix::Matrix(const Matrix & cpy)
{
	if (cpy.cpu_update) {
		cpy.updateGPU();
	}

	x_dim = cpy.x_dim;
	y_dim = cpy.y_dim;
	sz = cpy.sz;

	initialize();
	copy <<<n_blocks, n_threads >> > (gpu_matrix, cpy.gpu_matrix, sz);
	gpu_update = true;
}

Matrix::Matrix()
{
	x_dim = -1;
	y_dim = -1;
	sz = -1;
	initialized = false;
}

Matrix::~Matrix()
{
	hipFree(gpu_matrix);
	delete[] cpu_matrix;
}

int Matrix::length() const
{
	return x_dim;
}

int Matrix::width() const
{
	return y_dim;
}

int Matrix::size() const
{
	return sz;
}

void Matrix::print()
{
	//std::cout << " print method does not support transposed print " << std::endl;
	updateCPU();
		//std::cout << " not " << std::endl;
		for (int i = 0; i < sz; ++i) {
			if (i % x_dim == 0) {
				std::cout << std::endl;
			}
			std::cout << cpu_matrix[i] << " ";
		}
	std::cout << std::endl;
}


void Matrix::print(int p)
{
	std::cout.precision(p);
	print();
}
double & Matrix::operator[](int i)
{
	updateCPU();
	cpu_update = true;
	return cpu_matrix[i];
}

double & Matrix::operator()(int x, int y)
{
	//std::cout << " () oper method does not support transpose " << std::endl;

	updateCPU();
	cpu_update = true;
	return cpu_matrix[y * length() + x];
}

Matrix & Matrix::operator=(const Matrix & m)
{
	chkBounds(m);
	updateGPU();
	gpu_update = true;

	if (length() != m.length() || width() != m.width()) {
		throw std::invalid_argument("length/width mismatch");
	}

	copy<<<n_blocks, n_threads>>>(gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();
	return *this;
}

Matrix Matrix::T()
{
	return transpose();
}

Matrix Matrix::transpose() const
{
	//does not work
	Matrix r = Matrix(width(), length());

	Matrices_gpu::transpose << <n_blocks, n_threads >> >(r.gpu_matrix, gpu_matrix, length(), width());
	hipDeviceSynchronize();
	r.gpu_update = true;

	return r;
}

void Matrix::fill(double v)
{
	gpu_update = true;
	Matrices_gpu::fill <<<n_blocks, n_threads >>>(gpu_matrix, v, sz);
	hipDeviceSynchronize();
}

Matrix Matrix::operator^(const Matrix & m) const
{
	chkBounds(m);

	updateGPU();
	m.updateGPU();

	Matrix r = Matrix(x_dim, y_dim);
	exp << <n_blocks, n_threads >> > (r.gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;
	return r;
}

Matrix Matrix::operator*(const Matrix & m) const
{
	//Yeah this code sucks -- will try to make it pretty later 

	updateGPU();
	m.updateGPU();

	chkBounds_dotProduct(m);
	Matrix r = Matrix(width(), m.length());

	int n_blocks = 256; //arbitrary init
	int n_threads = 32; //arbitrary init
	int n_computations = length() * width() * m.width();

	//If there are only 1 set of threads 
	if (n_computations < 32) {
		n_blocks = 1;
		n_threads = 1;
	}
	else {
		n_blocks = ceil(n_computations / 32);
		n_threads = ceil(n_computations / n_blocks);

		if (n_blocks > MAX_BLOCK_dimx) {
			n_blocks = MAX_BLOCK_dimx;
	
			n_threads = ceil(n_computations / n_blocks);
			if (n_threads > MAX_THREADS) {
				n_threads = MAX_THREADS;
			}
		}
	}

	dot << <n_blocks, n_threads >> > (r.gpu_matrix, gpu_matrix, length(), width(), m.gpu_matrix, m.length(), m.width());
	hipDeviceSynchronize();

	r.gpu_update = true;
	return r;
}
Matrix Matrix::operator->*(const Matrix & m) const
{
	//Yeah this code sucks -- will try to make it pretty later 

	updateGPU();
	m.updateGPU();

	chkBounds_dotProduct_transposed(m);
	Matrix r = Matrix(width(), m.length());

	int n_blocks = 256; //arbitrary init
	int n_threads = 32; //arbitrary init
	int n_computations = length() * width() * m.width();

	//If there are only 1 set of threads 
	if (n_computations < 32) {
		n_blocks = 1;
		n_threads = 1;
	}
	else {
		n_blocks = ceil(n_computations / 32);
		n_threads = ceil(n_computations / n_blocks);

		if (n_blocks > MAX_BLOCK_dimx) {
			n_blocks = MAX_BLOCK_dimx;

			n_threads = ceil(n_computations / n_blocks);
			if (n_threads > MAX_THREADS) {
				n_threads = MAX_THREADS;
			}
		}
	}
	dot_transpose<< <n_blocks, n_threads >> > (r.gpu_matrix, gpu_matrix, x_dim, y_dim, m.gpu_matrix, m.length(), m.width());
	hipDeviceSynchronize();

	r.gpu_update = true;
	return r;
}
//LENGTH WIDTH X_DIM Y_DIM NOT UPDATED BELOW HERE
Matrix Matrix::operator&(const Matrix & m) const
{
	updateGPU();
	m.updateGPU(); 

	chkBounds(m);
	Matrix r = Matrix(length() , width());
	multiply << <n_blocks, n_threads >> > (r.gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;

	return r;
}

Matrix Matrix::operator/(const Matrix & m) const
{
	updateGPU();
	m.updateGPU();

	chkBounds(m);
	Matrix r = Matrix(length(), width());
	divide << <n_blocks, n_threads >> > (r.gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;

	return r;
}

Matrix Matrix::operator+(const Matrix & m) const
{
	updateGPU();
	m.updateGPU();

	chkBounds(m);
	Matrix r = Matrix(length(), width());
	add << <n_blocks, n_threads >> > (r.gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;

	return r;
}

Matrix Matrix::operator-(const Matrix & m) const
{
	updateGPU();
	m.updateGPU();

	chkBounds(m);
	Matrix r = Matrix(length(), width());
	subtract << <n_blocks, n_threads >> > (r.gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;

	return r;
}

Matrix& Matrix::operator^=(const Matrix & m)
{
	updateGPU();
	m.updateGPU();
	gpu_update = true;

	chkBounds(m);
	exp << <n_blocks, n_threads >> > (gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();

	return *this;
}

Matrix & Matrix::operator&=(const Matrix & m)
{
	chkBounds(m);
	updateGPU();
	m.updateGPU();
	gpu_update = true;

	multiply << <n_blocks, n_threads >> > (gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();
	return *this;
}

Matrix & Matrix::operator/=(const Matrix & m)
{
	chkBounds(m);
	updateGPU();
	m.updateGPU();
	gpu_update = true;

	divide << <n_blocks, n_threads >> > (gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();
	return *this;
}

Matrix & Matrix::operator+=(const Matrix & m)
{
	chkBounds(m);
	updateGPU();
	m.updateGPU();
	gpu_update = true;

	add << <256, 256 >> >(gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();
	return *this;
}

Matrix & Matrix::operator-=(const Matrix & m)
{
	chkBounds(m);
	updateGPU();
	m.updateGPU();
	gpu_update = true;

	subtract << <n_blocks, n_threads >> >(gpu_matrix, gpu_matrix, m.gpu_matrix, sz);
	hipDeviceSynchronize();
	return *this;
}

Matrix Matrix::operator^(double v) const
{
	updateGPU();

	Matrix r = Matrix(length(), width());
	exp << <n_blocks, n_threads >> >(r.gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;
	return r;
}

Matrix Matrix::operator/(double v) const
{
	updateGPU();

	Matrix r = Matrix(length(), width());
	divide << <n_blocks, n_threads >> >(r.gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;
	return r;
}

Matrix Matrix::operator+(double v) const
{
	updateGPU();

	Matrix r = Matrix(length(), width());
	add << <n_blocks, n_threads >> >(r.gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();
	
	r.gpu_update = true;
	return r;
}

Matrix Matrix::operator-(double v) const
{
	updateGPU();

	Matrix r = Matrix(length(), width());
	subtract << <n_blocks, n_threads >> >(r.gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;
	return r;
}

Matrix Matrix::operator&(double v) const
{
	updateGPU();

	Matrix r = Matrix(length(), width());
	multiply << <n_blocks, n_threads >> >(r.gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	r.gpu_update = true;
	return r;
}

Matrix & Matrix::operator^=(double v)
{
	updateGPU();
	gpu_update = true;

	exp << <n_blocks, n_threads >> >(gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	return *this;
}

Matrix & Matrix::operator/=(double v)
{
	updateGPU();
	gpu_update = true;

	divide << <n_blocks, n_threads >> >(gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	return *this;
}

Matrix & Matrix::operator+=(double v)
{
	updateGPU();
	gpu_update = true;

	add << <n_blocks, n_threads >> >(gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	return *this;
}

Matrix & Matrix::operator-=(double v)
{
	updateGPU();
	gpu_update = true;

	subtract << <n_blocks, n_threads >> >(gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	return *this;
}

Matrix & Matrix::operator&=(double v)
{
	updateGPU();
	gpu_update = true;

	multiply << <n_blocks, n_threads >> >(gpu_matrix, gpu_matrix, v, sz);
	hipDeviceSynchronize();

	return *this;
}

void Matrix::chkBounds(const Matrix & m) const
{
	if (length() != m.length()) {
		std::cout << "chkBounds fail *this.length() [" <<
			length() << "] != param.length() [" << m.length() << "]" << std::endl;
		throw std::invalid_argument("err");
	}
	else if (width() != m.width()) {
		std::cout << "chkBounds fail *this.width() [" <<
			width() << "] != param.width() [" << m.width() << "]" << std::endl;
		throw std::invalid_argument("err");
	}
}
void Matrix::chkBounds_dotProduct(const Matrix & m) const
{
	if (width() != m.length()) {
		std::cout << "chkBounds dotProduct fail this.width() = " << width() << " param.m.width() == " << m.width() << std::endl;
		throw std::invalid_argument("error");
	}
}

void Matrix::chkBounds_dotProduct_transposed(const Matrix & m) const
{
	if (length() != m.length()) {
		std::cout << "chkBounds dotProduct.T() fail this.length() = " << length() << " param.m.length() == " << m.length() << std::endl;
		throw std::invalid_argument("error");
	}
}
void Matrix::chkBounds_dot_dimensions(const Matrix & m1, const Matrix & m2) const
{
	m1.chkBounds_dotProduct(m2);
	if (length() != m1.length() || width() != m2.width()) {
		std::cout << "chkBounds dot product set dimensions invalid " << std::endl;
		throw std::invalid_argument("error");

	}
}
void Matrix::chkBounds(const Matrix & m, std::string message) const
{
	if (length() != m.length()) {
		std::cout << &message << std::endl;
		std::cout << "chkBounds fail *this.length() [" <<
			length() << "] != param.length() [" << m.length() << "]" << std::endl;
		throw std::invalid_argument("err");
	}
	else if (width() != m.width()) {
		std::cout << &message << std::endl;
		std::cout << "chkBounds fail *this.width() [" <<
			width() << "] != param.width() [" << m.width() << "]" << std::endl;
		throw std::invalid_argument("err");
	}
}
void Matrix::chkBounds_dotProduct(const Matrix & m, std::string message) const
{
	if (width() != m.length()) {
		std::cout << &message << std::endl;
		std::cout << "chkBounds dotProduct fail this.width() = " << width() << " param.m.width() == " << m.width() << std::endl;
		throw std::invalid_argument("error");
	}
}
void Matrix::chkBounds_dot_dimensions(const Matrix & m1, const Matrix & m2, std::string message) const
{
	m1.chkBounds_dotProduct(m2);
	if (length() != m1.length() || width() != m2.width()) {
		std::cout << &message << std::endl;
		std::cout << "chkBounds dot product set dimensions invalid " << std::endl;
		throw std::invalid_argument("error");
	}
}

