#include "hip/hip_runtime.h"
#include "GPU.h"

void initialize_blockThreads(unsigned& numb_blocks, unsigned& numb_threads, const unsigned* ranks, unsigned order)
{

	numb_blocks = 8;
	numb_threads = 512;

	return;

//	if (order > 1)
//		if (ranks[0] < 1024)
//		{
//			numb_threads = ranks[order - 1];
//			numb_blocks = 1;
//		} else
//		{
//			numb_threads = 128;
//			numb_blocks = ceil(ranks[order - 1] / numb_threads);
//
//		}
}

void initialize_blockThreads(unsigned& numb_blocks, unsigned& numb_threads, unsigned sz)
{
	numb_blocks = 8;
	numb_threads = 512;

	return;

//	if (sz < 1024)
//	{
//		numb_threads = sz;
//		numb_blocks = 1;
//	} else
//	{
//		numb_threads = 128;
//		numb_blocks = ceil(sz / numb_threads);
//
//	}
}

__global__
void one_init(float* f) {
	*f = 1;
}

void GPU::dot(float* store, unsigned store_LD, const float* m1, unsigned m1_row, unsigned m1_col, unsigned m1_inc, const float* m2, unsigned m2_row, unsigned m2_col, unsigned m2_inc)
{


	float* scal_one;
	hipMalloc((void**)&scal_one, sizeof(float));

	one_init<<<numb_blocks, numb_threads>>>(scal_one);
    hipblasHandle_t h;
    hipblasCreate(&h);
    hipblasSetPointerMode(h, HIPBLAS_POINTER_MODE_DEVICE);
	hipblasSgemm(h,
			HIPBLAS_OP_N,
			HIPBLAS_OP_N,
            m1_row, //height
            m2_col, //width
            m1_col,//wdith
            scal_one, //alpha
            m1, //d matb
            m1_inc, //matb ld

            m2, //d mat a
            m2_inc, //mat a ld
            scal_one, //beta scal
            store, //dmat x
            store_LD); //ld

	hipDeviceSynchronize();
	hipFree(scal_one);
	hipblasDestroy(h);
	hipDeviceSynchronize();

}

void GPU::max(float* max, const float* m1, const unsigned* ranks, const unsigned* ld, unsigned order)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, ranks, order);

	GPU_MATHEMATICS::max<<<numb_blocks, numb_threads>>>(max, m1, ranks, ld, order);
	hipDeviceSynchronize();
}
void GPU::min(float* min, const float* m1, const unsigned* ranks, const unsigned* ld, unsigned order)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, ranks, order);

	GPU_MATHEMATICS::min<<<numb_blocks, numb_threads>>>(min, m1, ranks, ld, order);
	hipDeviceSynchronize();
}

void GPU::min_index(float* min_val, unsigned* min_indexes, const float* data, const unsigned* ranks, const unsigned* ld, unsigned order)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, ranks, order);

	GPU_MATHEMATICS::min_index<<<numb_blocks, numb_threads>>>(min_val, min_indexes, data, ranks, ld, order);
	hipDeviceSynchronize();
}
void GPU::max_index(float* min_val, unsigned* min_indexes, const float* data, const unsigned* ranks, const unsigned* ld, unsigned order)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, ranks, order);

	GPU_MATHEMATICS::max_index<<<numb_blocks, numb_threads>>>(min_val, min_indexes, data, ranks, ld, order);
	hipDeviceSynchronize();
}

void GPU::correlation(float* s, const float* filter, const float* signal, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);

	GPU_MATHEMATICS::correlation<<<numb_blocks, numb_threads>>>(s, filter, signal ,sz);
	hipDeviceSynchronize();
}
void GPU::correlation(float* s, unsigned order, const unsigned* ranks, const float* filter, const unsigned* f_ld, const float* signal, const unsigned* s_ld)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, ranks, order);

	GPU_MATHEMATICS::correlation<<<numb_blocks, numb_threads>>>(s, order, ranks, filter, f_ld, signal, s_ld);
	hipDeviceSynchronize();
}
//dimensional --movement correlation

void GPU::cross_correlation(float* s, unsigned cor_mv, const unsigned* store_ld, const float* filter, const unsigned * f_ld, const unsigned* f_ranks, unsigned f_order, const float* signal, const unsigned * s_ld, const unsigned* s_ranks, unsigned s_order)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, s_order);

	GPU_MATHEMATICS::cross_correlation<<<numb_blocks, numb_threads>>>(s, cor_mv, store_ld, filter, f_ld, f_ranks, f_order, signal, s_ld, s_ranks, s_order);
	hipDeviceSynchronize();
}
void GPU::cross_correlation_filter_error(unsigned cor_mv, float* s, const unsigned* store_ld, const unsigned* store_ranks, unsigned store_order, const float* filter, const unsigned * f_ld, const unsigned* f_ranks, unsigned f_order, const float* signal, const unsigned * s_ld, const unsigned* s_ranks, unsigned s_order)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, s_order);

	GPU_MATHEMATICS::cc_filter_error<<<numb_blocks, numb_threads>>>(cor_mv, s, store_ld, store_ranks, store_order, filter, f_ld, f_ranks, f_order, signal, s_ld, s_ranks, s_order);
	hipDeviceSynchronize();
}

void GPU::cross_correlation_signal_error(unsigned cor_mv, float* s, const unsigned* store_ld, const unsigned* store_ranks, unsigned store_order, const float* filter, const unsigned * f_ld, const unsigned* f_ranks, unsigned f_order, const float* signal, const unsigned * s_ld, const unsigned* s_ranks, unsigned s_order)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, s_order);

	GPU_MATHEMATICS::cc_signal_error<<<numb_blocks, numb_threads>>>(cor_mv, s, store_ld, store_ranks, store_order, filter, f_ld, f_ranks, f_order, signal, s_ld, s_ranks, s_order);
	hipDeviceSynchronize();
}

//Memory Management [written for preparation for writing with CUDA]]
void GPU::initialize(float*& d, unsigned sz)
{
	GPU_MATHEMATICS::initialize(d, sz);
	hipDeviceSynchronize();

}
void GPU::initialize(unsigned*& d, unsigned sz)
{
	GPU_MATHEMATICS::initialize(d, sz);
	hipDeviceSynchronize();

}
void GPU::unified_initialize(float*& d, unsigned sz)
{
	GPU_MATHEMATICS::unified_initialize(d, sz);
	hipDeviceSynchronize();
}
void GPU::destruction(float* d)
{
	GPU_MATHEMATICS::destruction(d);
	hipDeviceSynchronize();
}

//unsgined
void GPU::unified_initialize(unsigned*& d, unsigned sz)
{
	GPU_MATHEMATICS::unified_initialize(d, sz);
	hipDeviceSynchronize();
}
void GPU::destruction(unsigned* d)
{
	GPU_MATHEMATICS::destruction(d);
	hipDeviceSynchronize();
}

void GPU::port_copy(float* s, const float* m, unsigned sz) {
	GPU_MATHEMATICS::port_copy(s, m, sz);
}

void GPU::port_copy_device_to_host(float* s, const float* m, unsigned sz) {
	GPU_MATHEMATICS::port_copy_device_to_host(s, m, sz);
}

void GPU::port_copy_device_to_host(unsigned* s, const unsigned* m, unsigned sz) {
	GPU_MATHEMATICS::port_copy_device_to_host(s, m, sz);
}
void GPU::copy(unsigned* s, const unsigned* m, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);

	GPU_MATHEMATICS::copy<<<numb_blocks, numb_threads>>>(s, m ,sz);
	hipDeviceSynchronize();
}
//end
void GPU::copy(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::copy<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD);
	hipDeviceSynchronize();
}

void GPU::copy(float* s, const float* m, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);

	GPU_MATHEMATICS::copy<<<numb_blocks, numb_threads>>>(s, m ,sz);
	hipDeviceSynchronize();
}

void GPU::fill(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, float m1)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::fill<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1);
	hipDeviceSynchronize();
}
void GPU::fill(float* s, float value, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);

	GPU_MATHEMATICS::fill<<<numb_blocks, numb_threads>>>(s, value, sz);
	hipDeviceSynchronize();
}
void GPU::randomize(float* s, float lower_bound, float upper_bound, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);

	GPU_MATHEMATICS::randomize<<<numb_blocks, numb_threads>>>(s, lower_bound, upper_bound, sz);
	hipDeviceSynchronize();
}

void GPU::transpose(float* s, unsigned s_ld, const float* m, unsigned r, unsigned c, unsigned m_ld)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, r * c);

	GPU_MATHEMATICS::transpose<<<numb_blocks, numb_threads>>>(s, s_ld, m, r, c, m_ld);
	hipDeviceSynchronize();
}

void GPU::power(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float* m2, const unsigned* m2_LD)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::power<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, m2, m2_LD);
	hipDeviceSynchronize();
}
void GPU::multiply(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float* m2, const unsigned* m2_LD)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::multiply<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, m2, m2_LD);
	hipDeviceSynchronize();
}
void GPU::divide(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float* m2, const unsigned* m2_LD)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::divide<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, m2, m2_LD);
	hipDeviceSynchronize();
}
void GPU::add(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float* m2, const unsigned* m2_LD)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::add<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, m2, m2_LD);
	hipDeviceSynchronize();
}
void GPU::subtract(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float* m2, const unsigned* m2_LD)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::subtract<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, m2, m2_LD);
	hipDeviceSynchronize();
}
//By Scalar
void GPU::power(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float scal)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::power<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, scal);
	hipDeviceSynchronize();
}
void GPU::multiply(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float scal)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::multiply<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, scal);
	hipDeviceSynchronize();
}
void GPU::divide(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float scal)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::divide<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, scal);
	hipDeviceSynchronize();
}
void GPU::add(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float scal)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::add<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, scal);
	hipDeviceSynchronize();
}
void GPU::subtract(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD, const float scal)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, s_ranks, order);

	GPU_MATHEMATICS::subtract<<<numb_blocks, numb_threads>>>(s, s_ranks, order, s_LD, m1, m1_LD, scal);
	hipDeviceSynchronize();
}
void GPU::print(const float* ary, const unsigned* dims, const unsigned* lead_dims, unsigned index)
{
	unsigned sz = 1;
	for (unsigned i = 0; i < index; ++i)
	{
		sz *= dims[i];
	}
	float* tmp = new float[sz];
	hipMemcpy(tmp, ary, sizeof(float) * sz, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	GPU_MATHEMATICS::print(tmp, dims, lead_dims, index);

	delete[] tmp;
	std::cout << " fin print " << std::endl;
}

void GPU::power(float* s, const float* m1, const float* m2, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);

	GPU_MATHEMATICS::power<<<numb_blocks, numb_threads>>>(s, m1, m2 ,sz);
	hipDeviceSynchronize();

}
void GPU::divide(float* s, const float* m1, const float* m2, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::divide<<<numb_blocks, numb_threads>>>(s, m1, m2 ,sz);
	hipDeviceSynchronize();


}
void GPU::add(float* s, const float* m1, const float* m2, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::add<<<numb_blocks, numb_threads>>>(s, m1, m2 ,sz);
	hipDeviceSynchronize();
}
void GPU::subtract(float* s, const float* m1, const float* m2, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::subtract<<<numb_blocks, numb_threads>>>(s, m1, m2 ,sz);
	hipDeviceSynchronize();

}
void GPU::multiply(float* s, const float* m1, const float* m2, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::multiply<<<numb_blocks, numb_threads>>>(s, m1, m2 ,sz);
	hipDeviceSynchronize();

}
//Pointwise Scalar
void GPU::power(float *s, const float *m, float v, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::power<<<numb_blocks, numb_threads>>>(s, m, v ,sz);
	hipDeviceSynchronize();

}
void GPU::divide(float *s, const float *m, float v, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::divide<<<numb_blocks, numb_threads>>>(s, m, v ,sz);
	hipDeviceSynchronize();

}
void GPU::add(float *s, const float *m, float v, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::add<<<numb_blocks, numb_threads>>>(s, m, v ,sz);
	hipDeviceSynchronize();

}
void GPU::subtract(float *s, const float *m, float v, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::subtract<<<numb_blocks, numb_threads>>>(s, m, v ,sz);
	hipDeviceSynchronize();

}
void GPU::multiply(float *s, const float *m, float v, unsigned sz)
{
	unsigned numb_blocks;
	unsigned numb_threads;
	initialize_blockThreads(numb_blocks, numb_threads, sz);
	GPU_MATHEMATICS::multiply<<<numb_blocks, numb_threads>>>(s, m, v ,sz);
	hipDeviceSynchronize();
}

