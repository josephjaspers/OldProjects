/*
 * Expression_Unary_Negation.h
 *
 *  Created on: Dec 26, 2017
 *      Author: joseph
 */
#ifdef  __HIPCC__
#ifndef EXPRESSION_UNARY_NEGATION_H_
#define EXPRESSION_UNARY_NEGATION_H_

#include "Expression_Base.cu"
namespace BC {
template<class, class >
class Array_Type;

template<class T, class functor_type>
struct unaryExpression_negation: expression<T, unaryExpression_negation<T, functor_type>> {

	functor_type array;

	unaryExpression_negation(functor_type cpy) :
			array(cpy) {
	}

	T operator [](int index) const {
		return -array[index];
	}
};

}

#endif /* EXPRESSION_UNARY_NEGATION_H_ */
#endif
