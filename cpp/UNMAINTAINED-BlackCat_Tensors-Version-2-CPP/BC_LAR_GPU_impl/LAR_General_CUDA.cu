#include "hip/hip_runtime.h"
/*
 * To change this license header, choose License Headers in Project Properties.
 * To change this template file, choose Tools | Templates
 * and open the template in the editor.
 */

/*
 * File:   LAR_General.h
 * Author: joseph
 *
 * Created on July 23, 2017, 3:24 PM
 */

#include "BLACKCAT_GPU_MATHEMATICS.cuh"


 void GPU_MATHEMATICS::initialize(float*& d, unsigned sz) {
	 hipMalloc((void**)&d, sizeof(float) * sz);
 }
 void GPU_MATHEMATICS::initialize(unsigned*& d, unsigned sz) {
	 hipMalloc((void**)&d, sizeof(unsigned) * sz);
 }
 void GPU_MATHEMATICS::unified_initialize(unsigned*& d, unsigned sz) {
	hipMallocManaged((void**)&d, sizeof(unsigned) * sz);
}
 void GPU_MATHEMATICS::unified_initialize(float*& d, unsigned sz) {
	hipMallocManaged((void**)&d, sizeof(float) * sz);
 }

 void GPU_MATHEMATICS::destruction(float * d) {
    hipFree(d);
    d = nullptr;
}
 void GPU_MATHEMATICS::destruction(unsigned * d) {
    hipFree(d);
    d = nullptr;
}

  void GPU_MATHEMATICS::port_copy(float * store, const float * v, unsigned sz) {
     hipMemcpy(store, v, sizeof(float) * sz, hipMemcpyHostToDevice);
 }
  void GPU_MATHEMATICS::port_copy_device_to_host(float * store, const float * v, unsigned sz) {
     hipMemcpy(store, v, sizeof(float) * sz, hipMemcpyDeviceToHost);
 }
  void GPU_MATHEMATICS::port_copy_device_to_host(unsigned * store, const unsigned * v, unsigned sz) {
     hipMemcpy(store, v, sizeof(unsigned) * sz, hipMemcpyDeviceToHost);
 }

__global__ void GPU_MATHEMATICS::copy(float * store, const float * v, unsigned sz) {
    for (int i = 0; i < sz; ++i) {
        store[i] = v[i];
    }
}
__global__ void GPU_MATHEMATICS::copy(unsigned * store, const unsigned * v, unsigned sz) {
    for (int i = 0; i < sz; ++i) {
        store[i] = v[i];
    }
}


__global__ void GPU_MATHEMATICS::fill(float * m, float f, unsigned sz) {
    for (int i = 0; i < sz; ++i) {
        m[i] = f;
    }
}


__global__ void GPU_MATHEMATICS::transpose(float * s, unsigned s_ld, const float * m, unsigned rows, unsigned cols, unsigned m_ld) {

	for (unsigned r = 0; r < rows; ++r) {
		for (unsigned c= 0; c< cols; ++c) {
			s[r * s_ld + c] = m[c * m_ld + r];
		}
	}
}


__global__ void GPU_MATHEMATICS::max(float* max_val, const float* data, const unsigned* ranks, const unsigned* ld, unsigned order) {

	if (order == 1) {
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			if (*max_val < data[i]) {
				*max_val = data[i];
			}
		}
	} else {

		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			max<<<128,128>>>(max_val, &data[i * ld[order-1]], ranks, ld, order-1);
		}
	}
}


__global__ void GPU_MATHEMATICS::min(float* min_val, const float* data, const unsigned* ranks, const unsigned* ld, unsigned order) {

	if (order == 1) {
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			if (*min_val < data[i]) {
				*min_val = data[i];
			}
		}
	} else {

		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			min<<<128,128>>>(min_val, &data[i * ld[order-1]], ranks, ld, order-1);
		}
	}
}


__global__ void GPU_MATHEMATICS::max_index(float* max_val, unsigned* max_indexes, const float* data, const unsigned* ranks, const unsigned* ld, unsigned order) {

	if (order == 1) {
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			if (*max_val < data[i]) {
				*max_val = data[i];
				*max_indexes = i;
			}
		}
	} else {

		float curr_max;
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			curr_max = *max_val;
			max_index<<<128,128>>>(max_val, max_indexes, &data[i * ld[order-1]], ranks, ld, order-1);

			//if modified -- update
			if (curr_max != *max_val) {
				max_indexes[order-1] = i;
			}
		}
	}
}



__global__ void GPU_MATHEMATICS::min_index(float* min_val, unsigned* min_indexes, const float* data, const unsigned* ranks, const unsigned* ld, unsigned order) {

	if (order == 1) {
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			if (*min_val < data[i]) {
				*min_val = data[i];
				*min_indexes = i;
			}
		}
	} else {

		float curr_max;
		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			curr_max = *min_val;
			min_index<<<128,128>>>(min_val, min_indexes, &data[i * ld[order-1]], ranks, ld, order-1);

			//if modified -- update
			if (curr_max != *min_val) {
				min_indexes[order-1] = i;
			}
		}
	}
}




__global__ void GPU_MATHEMATICS::randomize(float * m, float lower_bound, float upper_bound, unsigned sz) {
    for (int i = 0; i < sz; ++i) {
      if (i > 5) {
    	  m[i] = i;
      } else {
    	  m[i] = -i;
      }
    }
}


__global__ void GPU_MATHEMATICS::randomize(float * m, unsigned* ld, unsigned* ranks, unsigned order, float lower_bound, float upper_bound) {
    --order;
    if (order == 0) {
    	randomize<<<128,128>>>(m, lower_bound, upper_bound , ranks[order]);
    } else {
    	for (unsigned i = 0; i < ranks[order]; ++i) {
    		randomize<<<128,128>>>(&m[ld[order] * i], ld, ranks, order, lower_bound, upper_bound);
    	}
    }
}

void GPU_MATHEMATICS::print(const float* ary, unsigned sz) {
	for (unsigned i = 0; i < sz; ++i) {
		std::cout << ary[i] << " " << std::endl;
	}
}


 void GPU_MATHEMATICS::print(const float* ary, const unsigned* dims, const unsigned* lead_dims, unsigned index) {


	if (index < 3) {
		for (unsigned r = 0; r < dims[0]; ++r) {

			if (r != 0)
			std::cout << std::endl;

			for (unsigned c = 0; c< dims[1]; ++c) {
				auto str =std::to_string(ary[r + c * lead_dims[index - 1]]);
				str = str.substr(0, str.length() < 3 ? str.length() : 3);
				std::cout << str << " ";
			}
		}
		std::cout << "]" << std::endl << std::endl;

	} else {
		std::cout << "[";
		for (unsigned i = 0; i < dims[index - 1]; ++i) {
			print(&ary[i * lead_dims[index - 1]], dims, lead_dims, index - 1);
		}
	}
}
