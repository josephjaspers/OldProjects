#include "hip/hip_runtime.h"
#include "BLACKCAT_GPU_MATHEMATICS.cuh"


 __global__
void GPU_MATHEMATICS::power(float* s, const float* m1, const float* m2, unsigned sz)
{
	//NOT WRITTEN
}
 __global__
void GPU_MATHEMATICS::divide(float* s, const float* m1, const float* m2, unsigned sz)
{
	for (unsigned i = 0; i < sz; ++i)
	{
		s[i] = m1[i] / m2[i];
	}
}
 __global__
void GPU_MATHEMATICS::add(float* s, const float* m1, const float* m2, unsigned sz)
{

	for (unsigned i = 0; i < sz; ++i)
	{
		s[i] = m1[i] + m2[i];
	}
}
 __global__
void GPU_MATHEMATICS::subtract(float* s, const float* m1, const float* m2, unsigned sz)
{
	for (unsigned i = 0; i < sz; ++i)
	{
		s[i] = m1[i] - m2[i];
	}
}
 __global__
void GPU_MATHEMATICS::multiply(float* s, const float* m1, const float* m2, unsigned sz)
{
	for (unsigned i = 0; i < sz; ++i)
	{
		s[i] = m1[i] * m2[i];
	}
}
////Pointwise Scalar
 __global__
void GPU_MATHEMATICS::power(float *s, const float *m, float v, unsigned sz)
{
	 //NOT WRITTEN
}
 __global__
void GPU_MATHEMATICS::divide(float *s, const float *m, float v, unsigned sz)
{
	for (unsigned i = 0; i < sz; ++i)
	{
		s[i] = m[i] / v;
	}
}
 __global__
void GPU_MATHEMATICS::add(float *s, const float *m, float v, unsigned sz)
{
	for (unsigned i = 0; i < sz; ++i)
	{
		s[i] = m[i] + v;
	}
}
 __global__
void GPU_MATHEMATICS::subtract(float *s, const float *m, float v, unsigned sz)
{
	for (unsigned i = 0; i < sz; ++i)
	{
		s[i] = m[i] - v;
	}
}
 __global__
void GPU_MATHEMATICS::multiply(float *s, const float *m, float v, unsigned sz)
{
	for (unsigned i = 0; i < sz; ++i)
	{
		s[i] = m[i] * v;
	}
}

