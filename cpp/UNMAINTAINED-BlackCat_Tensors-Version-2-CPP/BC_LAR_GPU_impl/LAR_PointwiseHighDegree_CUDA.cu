#include "hip/hip_runtime.h"
#include "BLACKCAT_GPU_MATHEMATICS.cuh"



    __global__ void GPU_MATHEMATICS::copy(float* s, const unsigned* ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD) {

	unsigned store_index = 0;
	unsigned m1_index = 0;



	if (order  == 1) {
		copy<<<10,128>>>(s, m1, ranks[order-1]);

	} else {

		for (unsigned i = 0; i < ranks[order-1]; ++i) {
			copy<<<10,128>>>(&s[store_index], ranks, order-1, s_LD, &m1[m1_index], m1_LD);
			store_index += s_LD[order-1];
			m1_index += m1_LD[order-1];
		}
	}
}

    __global__ void GPU_MATHEMATICS::fill(float* s, const unsigned* s_ranks,  unsigned order, const unsigned *s_LD, float m1) {
	if (order == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1;
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			fill<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, m1);
		}
	}
}


//----

    __global__ void GPU_MATHEMATICS::power(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD,
    																															 const float* m2, const unsigned* m2_LD) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			//s[i] = pow(m1[i], m2[i]);
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			power<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
		}
	}
}


    __global__ void GPU_MATHEMATICS::multiply(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD,
    																															 const float* m2, const unsigned* m2_LD) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1[i] * m2[i];
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			multiply<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
		}
	}
}


    __global__ void GPU_MATHEMATICS::divide(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD,
    																															 const float* m2, const unsigned* m2_LD) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1[i] / m2[i];
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			divide<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
		}
	}
}


    __global__ void GPU_MATHEMATICS::add(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD,
    																															 const float* m2, const unsigned* m2_LD) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1[i] + m2[i];
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			add<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
		}
	}
}



    __global__ void GPU_MATHEMATICS::subtract(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1, const unsigned* m1_LD,
    																															 const float* m2, const unsigned* m2_LD) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1[i] - m2[i];
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			subtract<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
		}
	}
}

//-----------------------------------------------Scalar Methods ----------------------------------------------------//

__global__ void GPU_MATHEMATICS::power	(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD,
																			const float* m1,  const unsigned* m1_LD, const float scal) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
		//	s[i] = pow(m1[i], scal);
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			power<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
		}
	}
}

__global__ void GPU_MATHEMATICS::multiply(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD,
																			const float* m1,  const unsigned* m1_LD, const float scal) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1[i] * scal;
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			multiply<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
		}
	}
}

__global__ void GPU_MATHEMATICS::divide(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1,
																			const unsigned* m1_LD, const float scal) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1[i] / scal;
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			divide<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
		}
	}
}

__global__ void GPU_MATHEMATICS::add(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1,
																			const unsigned* m1_LD, const float scal) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1[i] + scal;
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			add<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
		}
	}
}

__global__ void GPU_MATHEMATICS::subtract(float* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const float* m1,
																			const unsigned* m1_LD, const float scal) {
	if (order  == 1) {
		for (unsigned i = 0; i < s_ranks[0]; ++i) {
			s[i] = m1[i] - scal;
		}
	} else {
		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
			subtract<<<10,128>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
		}
	}
}

