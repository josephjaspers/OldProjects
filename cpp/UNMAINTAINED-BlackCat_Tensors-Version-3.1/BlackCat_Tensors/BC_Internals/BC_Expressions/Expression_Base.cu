/*

 * BC_Expression_Base.h
 *
 *  Created on: Dec 11, 2017
 *      Author: joseph
 */

#ifdef  __HIPCC__
#ifndef EXPRESSION_BASE_H_
#define EXPRESSION_BASE_H_
namespace BC {

#include <hip/hip_runtime.h>
#include "../BlackCat_Internal_Definitions.h"


template<class T, class derived>
struct expression {
	using type = derived;
	using scalar_type = T;
};

template<class T, class derived>
struct non_linear_expression : expression<T, derived> {
	using type = derived;
	using scalar_type = T;
};



}

#endif /* EXPRESSION_BASE_H_ */
#endif
