
#include <hip/hip_runtime.h>
//#include "BLACKCAT_GPU_MATHEMATICS.cuh"
//
//
//
//    __global__ void GPU_MATHEMATICS::copy(unsigned* s, const unsigned* ranks, unsigned order, const unsigned *s_LD, const unsigned* m1, const unsigned* m1_LD) {
//
//	unsigned store_index = 0;
//	unsigned m1_index = 0;
//
//	if (order  == 1) {
//		copy<<<256,256>>>(s, m1, ranks[order-1]);
//
//	} else {
//
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			copy<<<256,256>>>(&s[store_index], ranks, order-1, s_LD, &m1[m1_index], m1_LD);
//			store_index += s_LD[order-1];
//			m1_index += m1_LD[order-1];
//		}
//	}
//}
//
//    __global__ void GPU_MATHEMATICS::fill(unsigned* s, const unsigned* s_ranks,  unsigned order, const unsigned *s_LD, unsigned m1) {
//	if (order == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1;
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			fill<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, m1);
//		}
//	}
//}
//
//
////----
//
//    __global__ void GPU_MATHEMATICS::power(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const unsigned* m1, const unsigned* m1_LD,
//    																															 const unsigned* m2, const unsigned* m2_LD) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			//s[i] = pow(m1[i], m2[i]);
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			power<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
//		}
//	}
//}
//
//
//    __global__ void GPU_MATHEMATICS::multiply(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const unsigned* m1, const unsigned* m1_LD,
//    																															 const unsigned* m2, const unsigned* m2_LD) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1[i] * m2[i];
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			multiply<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
//		}
//	}
//}
//
//
//    __global__ void GPU_MATHEMATICS::divide(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const unsigned* m1, const unsigned* m1_LD,
//    																															 const unsigned* m2, const unsigned* m2_LD) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1[i] / m2[i];
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			divide<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
//		}
//	}
//}
//
//
//    __global__ void GPU_MATHEMATICS::add(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const unsigned* m1, const unsigned* m1_LD,
//    																															 const unsigned* m2, const unsigned* m2_LD) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1[i] + m2[i];
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			add<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
//		}
//	}
//}
//
//
//
//    __global__ void GPU_MATHEMATICS::subtract(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const unsigned* m1, const unsigned* m1_LD,
//    																															 const unsigned* m2, const unsigned* m2_LD) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1[i] - m2[i];
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			subtract<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, &m2[m2_LD[order - 1] * i], m2_LD);
//		}
//	}
//}
//
////-----------------------------------------------Scalar Methods ----------------------------------------------------//
//
//__global__ void GPU_MATHEMATICS::power	(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD,
//																			const unsigned* m1,  const unsigned* m1_LD, const unsigned scal) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//		//	s[i] = pow(m1[i], scal);
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			power<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
//		}
//	}
//}
//
//__global__ void GPU_MATHEMATICS::multiply(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD,
//																			const unsigned* m1,  const unsigned* m1_LD, const unsigned scal) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1[i] * scal;
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			multiply<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
//		}
//	}
//}
//
//__global__ void GPU_MATHEMATICS::divide(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const unsigned* m1,
//																			const unsigned* m1_LD, const unsigned scal) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1[i] / scal;
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			divide<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
//		}
//	}
//}
//
//__global__ void GPU_MATHEMATICS::add(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const unsigned* m1,
//																			const unsigned* m1_LD, const unsigned scal) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1[i] + scal;
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			add<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
//		}
//	}
//}
//
//__global__ void GPU_MATHEMATICS::subtract(unsigned* s, const unsigned* s_ranks, unsigned order, const unsigned *s_LD, const unsigned* m1,
//																			const unsigned* m1_LD, const unsigned scal) {
//	if (order  == 1) {
//		for (unsigned i = 0; i < s_ranks[0]; ++i) {
//			s[i] = m1[i] - scal;
//		}
//	} else {
//		for (unsigned i = 0; i < s_ranks[order - 1]; ++i) {
//			subtract<<<256,256>>>(&s[s_LD[order - 1] * i], s_ranks, order - 1, s_LD, &m1[m1_LD[order - 1] * i], m1_LD, scal);
//		}
//	}
//}
//
