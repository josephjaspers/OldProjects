#include "hip/hip_runtime.h"
#include <iostream>
#include "../BlackCat_Tensors.h"
#include "SpeedTests.h"
using BC::Vector;
using BC::Matrix;
using BC::Scalar;

template<class T>
T copy(const T& t) { return t; }

int VectorPointwise() {

std::cout << "Vector test " <<std::endl;

	Vector<double> a = {5};
	Vector<double> b = {5};
	Vector<double> c = {5};

	std::cout << " getting size" <<std::endl;


	std::cout << b.size() << " getting size" <<std::endl;
	b.printDimensions();
//	a.printDimensions();

//	a.printDetails();


//	b.print();
	std::cout << "Randomizing " << std::endl;
	b.randomize(-3, 5);
	c.randomize(0, 5);

	std::cout << " B " << std::endl;
	b.print();
	std::cout << " C" << std::endl;
	c.print();


	std::cout << " c= b.t" << std::endl;

	c = b.t();
	c.print();

	std::cout << " B + C" << std::endl;
	a = b + c;
	a.print();

	std::cout << " B - C" << std::endl;
	a = b - c;


	std::cout << " B ** C" << std::endl;
	b.print();
	c.print();

	a = b ** c;
	a.print();


	std::cout << "b[0]" << std::endl;
	b[0].print();


	std::cout << "a" << std::endl;
	a.print();


	std::cout << "a += b[0]" << std::endl;
	a += b[0];
	a.print();


	std::cout << " success " << std::endl;
	return 0;


}

int MatrixPointwise() {

std::cout << "Matrix test " <<std::endl;

	Matrix<double> a(4,3);
	Matrix<double> b(4,3);
	Matrix<double> c(4,3);


	Matrix<double> z(5,5 );
	Vector<double> d(3);


	*d;
	Vector<double> e(4);

	b.randomize(0, 10);
	c.randomize(0, 10);

	b.print();
	c.print();

	std::cout << " c= b.t" << std::endl;

	c = b.t();

	std::cout << "A = b + c" << std::endl;
	a = b + c;

	a.print();



	a[0][1].print();

	std::cout << " adding array " << std::endl;
b.print();
b += a[0][1];
b.print();
c.printDetails();
d.printDetails();
	e = (c * d);


	e.print();
	e[1] = 2;
	z.randomize(1,2);

	e[1].print();

	z = z.t() * e[1] * z;

	z.print();
	std::cout << " success " << std::endl;

	return 0;
}


struct sigmoid {
	template<class T>
	T operator ()(T val) const {
		std::cout << " g func  " << std::endl;

		return 1/ (1 + std::pow(2.71828, -val));
	}
};



auto dp_test () {

	Matrix<double> a(3, 2);
	Matrix<double> b(2 ,3);

	for (int i = 0; i < 6; ++i)  {
		b.data()[i] = i + 7;
		a.data()[i] = i + 1;
	}

	std::cout << std::endl;

	Matrix<double> d(2, 3);
	Matrix<double> e(3, 2);



	d = a.t();
	e = b.t();


	a.print();
	b.print();
	d.print();
	e.print();


	std::cout << " post print  " << std::endl;
	Matrix<double> c(2, 2);

	Scalar<double> C(2);


	std::cout << " simple dot product " << std::endl;
	c = d * e;
	c.print();
	Scalar<double> A(2);
	Scalar<double> B(2);

	A.print();

	std::cout << " dot product " << std::endl;

	c = a.t() * A * (b.t() * B);
	std::cout << " dot product 222 " << std::endl;

	c = a.t() * A * (b.t() * Scalar<double>(2));
//
	c.print();

	std::cout << " copy " << std::endl;
	a.print();
	b.print();

	auto tmp = a + b;
//	Vector<double> c_copy = copy(tmp);
//	(a + b).printExpression();
	std::cout << "done  copy " << std::endl;
//	c_copy.print();


//	c.print();
//	c = g(c);
//	c.print();
////
//	c_copy.print();

	std::cout << "successsasd " << std::endl;
//	return 0;
//	return c_copy;
}


auto dp_test_gpu() {

	Matrix<float> a(3, 2);
	Matrix<float> b(2 ,3);

	Matrix<float> a_(2, 3);
	Matrix<float> b_(3 ,2);
	for (int i = 0; i < 6; ++i)  {
		b.data()[i] = i + 7;
		a.data()[i] = i + 1;
	}
	a_ = a.t();
	b_ = b.t();

	std::cout << std::endl;

	Matrix<float, BC::GPU> d(2, 3);
	Matrix<float, BC::GPU> e(3, 2);

	BC::GPU::HostToDevice(d.data(), a_.data(), 6);
	BC::GPU::HostToDevice(e.data(), b_.data(), 6);

	d.print();
	e.print();

	std::cout << " post print  " << std::endl;
	Matrix<float,BC::GPU> c(2, 2);

	Scalar<float,BC::GPU> C(2);


	std::cout << " simple dot product " << std::endl;
	c = d * e;
	c.print();
	Scalar<float,BC::GPU> A(2);
	Scalar<float,BC::GPU> B(2);

	A.print();

	std::cout << " dot product " << std::endl;
//
	c = d * A * (e * B);
	std::cout << " dot product 222 " << std::endl;

	c = d * A * (e * Scalar<float,BC::GPU>(2));

	c.print();

//	std::cout << " copy " << std::endl;
//	a.print();
//	b.print();
//
//	auto tmp = a + b;
//	Vector<double> c_copy = copy(tmp);
//	(a + b).printExpression();
//	std::cout << "done  copy " << std::endl;
//	c_copy.print();


//	c.print();
//	c = g(c);
//	c.print();
////
//	c_copy.print();

	std::cout << "successsasd " << std::endl;
//	return 0;
////	return c_copy;
}



auto dp_test_gpuM() {

	Matrix<float> a(3, 2);
	Matrix<float> b(2 ,3);

	Matrix<float> a_(2, 3);
	Matrix<float> b_(3 ,2);
	for (int i = 0; i < 6; ++i)  {
		b.data()[i] = i + 7;
		a.data()[i] = i + 1;
	}
	a_ = a.t();
	b_ = b.t();

	std::cout << std::endl;

	Matrix<float, BC::GPUM> d(2, 3);
	Matrix<float, BC::GPUM> e(3, 2);

	BC::GPU::HostToDevice(d.data(), a_.data(), 6);
	BC::GPU::HostToDevice(e.data(), b_.data(), 6);

	d.print();
	e.print();

	std::cout << " post print  " << std::endl;
	Matrix<float,BC::GPUM> c(2, 2);

	Scalar<float,BC::GPUM> C(2);


	std::cout << " simple dot product " << std::endl;
	c = d * e;
	c.print();
	Scalar<float,BC::GPUM> A(2);
	Scalar<float,BC::GPUM> B(2);

	A.print();

	std::cout << " dot product " << std::endl;
//
	c = d * A * (e * B);
	std::cout << " dot product 222 " << std::endl;

	c = d * A * (e * Scalar<float,BC::GPUM>(2));

	c.print();

//	std::cout << " copy " << std::endl;
//	a.print();
//	b.print();
//
//	auto tmp = a + b;
//	Vector<double> c_copy = copy(tmp);
//	(a + b).printExpression();
//	std::cout << "done  copy " << std::endl;
//	c_copy.print();


//	c.print();
//	c = g(c);
//	c.print();
////
//	c_copy.print();

	std::cout << "successsasd " << std::endl;
//	return 0;
////	return c_copy;
}


int readWrite() {

	std::ofstream os("input.txt");

	Matrix<double> a(3, 2);
	Matrix<double> b(2 ,3);

	for (int i = 0; i < 6; ++i)  {
		a.data()[i] = i + 7;
		b.data()[i] = i + 1;
	}

	a.print();
	b.print();

	a.write(os);
	b.write(os);
//	std::cout << std::endl;

	os.close();

	std::ifstream is("input.txt");

	Matrix<double> d(3, 2);
	Matrix<double> e(2, 3);


	d.read(is);
	e.read(is);

	is.close();
	std::cout << " read "<< std::endl;

	d.print();
	e.print();
	std::cout << " success "<< std::endl;
	return 0;
}

int main() {
//	gpu();
//	speedTests<8000000, 100000>();
//	speedTests<4000000, 10000>();
//	speedTests<2000000, 10000>();
//
//	speedTestCombine<128,     100000>();
//	speedTestCombine<256,     100000>();
//	speedTestCombine<512,     100000>();
//	speedTestCombine<1024,    100000>();
//	speedTestCombine<2048,    100000>();
//	speedTestCombine<5096,    100000>();
//	speedTestCombine<10000,   100000>();
//	speedTestCombine<20000,   100000>();
//	speedTestCombine<40000,   100000>();
//	speedTestCombine<80000,   100000>();
//	speedTestCombine<100000,  100000>();

//	speedTests<64,     100000>();
//	speedTests<32,  100000>();
//	speedTests<16,  100000>();
//	speedTests<8,   100000>();
//	speedTests<4,    100000>();
//	speedTests<2,     100000>();

//	c.printDimensions();
//	c.print();
//	readWrite();
//	VectorPointwise();
//	MatrixPointwise();
//	AND_example();
	dp_test_gpu();
	dp_test_gpuM();

	//	dp_test();



	std::cout << " success  main"<< std::endl;


	return 0;
}
