/*
 * Expression_Unary_Pointwise.cu
 *
 *  Created on: Jan 25, 2018
 *      Author: joseph
 */
#ifdef  __HIPCC__
#ifndef EXPRESSION_UNARY_POINTWISE_CU_
#define EXPRESSION_UNARY_POINTWISE_CU_

#include "Expression_Base.cu"
namespace BC {
template<class T, class operation, class value>
class unary_expression : public expression<T, unary_expression<T, operation, value>> {
public:

	using this_type = unary_expression<T, operation, value>;
	operation oper;

	value data;

	inline __attribute__((always_inline)) __BC_gcpu__ unary_expression(value v) :
			data(v) {
	}
	inline __attribute__((always_inline)) __BC_gcpu__ auto operator [](int index) const {
		return oper(data[index]);
	}
};

}



#endif /* EXPRESSION_UNARY_POINTWISE_CU_ */
#endif
