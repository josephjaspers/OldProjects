
#include <hip/hip_runtime.h>
///*
// * To change this license header, choose License Headers in Project Properties.
// * To change this template file, choose Tools | Templates
// * and open the template in the editor.
// */
//
///*
// * File:   LAR_General.h
// * Author: joseph
// *
// * Created on July 23, 2017, 3:24 PM
// */
//
//#include "BLACKCAT_GPU_MATHEMATICS.cuh"
//
////unsigned
////__global__ void GPU_MATHEMATICS::initialize(unsigned*& d, unsigned sz) {
////    d = new unsigned[sz];
////}
////
////unsigned
////__global__ void GPU_MATHEMATICS::destruction(unsigned * d) {
////    delete [] d;
////}
//
//
//__global__ void GPU_MATHEMATICS::copy(unsigned * store, const unsigned * v, unsigned sz) {
//    for (int i = 0; i < sz; ++i) {
//        store[i] = v[i];
//    }
//}
//
//
//__global__ void GPU_MATHEMATICS::fill(unsigned * m, unsigned f, unsigned sz) {
//    for (int i = 0; i < sz; ++i) {
//        m[i] = f;
//    }
//}
//
//
//__global__ void GPU_MATHEMATICS::transpose(unsigned * s, unsigned s_ld, const unsigned * m, unsigned rows, unsigned cols, unsigned m_ld) {
//
//	for (unsigned r = 0; r < rows; ++r) {
//		for (unsigned c= 0; c< cols; ++c) {
//			s[r * s_ld + c] = m[c * m_ld + r];
//		}
//	}
//}
//
//
//__global__ void GPU_MATHEMATICS::max(unsigned* max_val, const unsigned* data, const unsigned* ranks, const unsigned* ld, unsigned order) {
//
//	if (order == 1) {
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			if (*max_val < data[i]) {
//				*max_val = data[i];
//			}
//		}
//	} else {
//
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			max<<<256,256>>>(max_val, &data[i * ld[order-1]], ranks, ld, order-1);
//		}
//	}
//}
//
//
//__global__ void GPU_MATHEMATICS::min(unsigned* min_val, const unsigned* data, const unsigned* ranks, const unsigned* ld, unsigned order) {
//
//	if (order == 1) {
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			if (*min_val < data[i]) {
//				*min_val = data[i];
//			}
//		}
//	} else {
//
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			min<<<256,256>>>(min_val, &data[i * ld[order-1]], ranks, ld, order-1);
//		}
//	}
//}
//
//
//__global__ void GPU_MATHEMATICS::max_index(unsigned* max_val, unsigned* max_indexes, const unsigned* data, const unsigned* ranks, const unsigned* ld, unsigned order) {
//
//	if (order == 1) {
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			if (*max_val < data[i]) {
//				*max_val = data[i];
//				*max_indexes = i;
//			}
//		}
//	} else {
//
//		unsigned curr_max;
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			curr_max = *max_val;
//			max_index<<<256,256>>>(max_val, max_indexes, &data[i * ld[order-1]], ranks, ld, order-1);
//
//			//if modified -- update
//			if (curr_max != *max_val) {
//				max_indexes[order-1] = i;
//			}
//		}
//	}
//}
//
//
//
//__global__ void GPU_MATHEMATICS::min_index(unsigned* min_val, unsigned* min_indexes, const unsigned* data, const unsigned* ranks, const unsigned* ld, unsigned order) {
//
//	if (order == 1) {
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			if (*min_val < data[i]) {
//				*min_val = data[i];
//				*min_indexes = i;
//			}
//		}
//	} else {
//
//		unsigned curr_max;
//		for (unsigned i = 0; i < ranks[order-1]; ++i) {
//			curr_max = *min_val;
//			min_index<<<256,256>>>(min_val, min_indexes, &data[i * ld[order-1]], ranks, ld, order-1);
//
//			//if modified -- update
//			if (curr_max != *min_val) {
//				min_indexes[order-1] = i;
//			}
//		}
//	}
//}
//
//
//
//
//__global__ void GPU_MATHEMATICS::randomize(unsigned * m, unsigned lower_bound, unsigned upper_bound, unsigned sz) {
//    for (int i = 0; i < sz; ++i) {
//        m[i] =  i % 2 == 0 ? 3 : -3 ;
//    }
//}
//
//
//__global__ void GPU_MATHEMATICS::randomize(unsigned * m, unsigned* ld, unsigned* ranks, unsigned order, unsigned lower_bound, unsigned upper_bound) {
//    --order;
//    if (order == 0) {
//    	randomize<<<256,256>>>(m, lower_bound, upper_bound , ranks[order]);
//    } else {
//    	for (unsigned i = 0; i < ranks[order]; ++i) {
//    		randomize<<<256,256>>>(&m[ld[order] * i], ld, ranks, order, lower_bound, upper_bound);
//    	}
//    }
//}
//
//void GPU_MATHEMATICS::print(const unsigned* ary, unsigned sz) {
//	for (unsigned i = 0; i < sz; ++i) {
//		std::cout << ary[i] << " " << std::endl;
//	}
//}
//
//
//__global__ void GPU_MATHEMATICS::print(const unsigned* ary, const unsigned* dims, const unsigned* lead_dims, unsigned index) {
//
//
////	if (index < 3) {
////		for (unsigned r = 0; r < dims[0]; ++r) {
////
////			if (r != 0)
////			std::cout << std::endl;
////
////			for (unsigned c = 0; c< dims[1]; ++c) {
////				auto str =std::to_string(ary[r + c * lead_dims[index - 1]]);
////				str = str.substr(0, str.length() < 3 ? str.length() : 3);
////				std::cout << str << " ";
////			}
////		}
////		std::cout << "]" << std::endl << std::endl;
////
////	} else {
////		std::cout << "[";
////		for (unsigned i = 0; i < dims[index - 1]; ++i) {
////			print(&ary[i * lead_dims[index - 1]], dims, lead_dims, index - 1);
////		}
////	}
//}
