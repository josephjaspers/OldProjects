/*
 * Expression_Binary_Dotproduct_impl2.h
 *
 *  Created on: Jan 23, 2018
 *      Author: joseph
 */
#ifdef  __HIPCC__
#ifndef EXPRESSION_BINARY_DOTPRODUCT_IMPL2_H_
#define EXPRESSION_BINARY_DOTPRODUCT_IMPL2_H_

#include "../BC_MetaTemplateFunctions/Simple.h"
#include "../BC_MetaTemplateFunctions/Adhoc.h"

#include <type_traits>
#include "Expression_Binary_Functors.cu"

namespace BC {

template<class, class, class, class > class binary_expression_scalar_R;
template<class, class, class, class > class binary_expression_scalar_L;
template<class, class > class unary_expression_transpose;
template<class, class, class, class> class binary_expression;
template<class, class > class expression;
class mul;



/*
 * a = M x K
 * b = K x N
 * c = M x N
 */

template<class> struct self_eval { static constexpr bool conditional = false; };
template<class T, class lv, class rv> struct self_eval<binary_expression<T, assign, lv, rv>> { static constexpr bool conditional = true; };

//conditional indicates whether or not the array needs to be evaluated
template<class T> struct add_ptr { using type = T*; };

//Generic (Unknown Expression)
template<class T, class enabler = void> struct evaluate {
	static constexpr bool conditional = true;
		static constexpr bool transpose = false;
		static constexpr bool scalar = false;

		template<typename param> static typename add_ptr<typename MTF::determine_scalar<T>::type>::type getScalar(const param& p) {
			return nullptr;
		}
		template<typename param> static auto getArray(const param& p) {
			return p;
		}
};
//Standard Functor_type
template<class T> struct evaluate<T*> {
	static constexpr bool conditional = false;
	static constexpr bool transpose = false;
	static constexpr bool scalar = false;

	template<typename param> static T* getScalar(const param& p) {
		return nullptr;
	}
	template<typename param> static auto getArray(const param& p) {
		return p;
	}
};

template<class T, class U>
struct evaluate<unary_expression_transpose<T, U*>> {

	static constexpr bool conditional = false;
	static constexpr bool transpose = true;
	static constexpr bool scalar = false;
	template<typename param> static T* getScalar(const param& p) {
		return nullptr;
	}
	template<typename param> static auto getArray(const param& p) {
		return p.array;
	}
};

template<class T, class lv, class rv>
struct evaluate<binary_expression<T, assign, lv, rv>> {

	static constexpr bool conditional = true;
	static constexpr bool transpose = false;
	static constexpr bool scalar = false;
	template<typename param> static T* getScalar(const param& p) {
		return nullptr;
	}
	template<typename param> static auto getArray(const param& p) {
		return p;
	}
};


//Simple Scalar by tensor R
template<class T, class lv, class rv> struct evaluate<
		binary_expression_scalar_R<T, mul, lv*, rv*>> {
	static constexpr bool conditional = false;
	static constexpr bool transpose = false;
	static constexpr bool scalar = true;

	template<typename param> static auto getScalar(const param& p) {
		return p.right;
	}
	template<typename param> static auto getArray(const param& p) {
		return p.left;
	}
};
//Simple Scalar by tensor L
template<class T, class lv, class rv> struct evaluate<
		binary_expression_scalar_L<T, mul, lv*, rv*>> {
	static constexpr bool conditional = false;
	static constexpr bool transpose = false;
	static constexpr bool scalar = true;

	template<typename param> static auto getScalar(const param& p) {
		return p.left;
	}
	template<typename param> static auto getArray(const param& p) {
		return p.right;
	}
};

//scalar R with unary transposition on inside
template<class T, class U, class rv, class V>
struct evaluate<
		binary_expression_scalar_R<T, mul,
				unary_expression_transpose<U, V*>, rv*>> { //change tensor to enable only if primitive

	static constexpr bool conditional = false;
	static constexpr bool transpose = true;
	static constexpr bool scalar = true;

	template<typename param> static auto getScalar(const param& p) {
		return p.right;
	}
	template<typename param> static auto getArray(const param& p) {
		return p.left.array;
	}

};

//scalar L with unary transposition on inside
template<class T, class U, class lv, class V>
struct evaluate<
		binary_expression_scalar_L<T, mul, lv*,
				unary_expression_transpose<U, V*>>> { //change tensor to enable only if primitive

	static constexpr bool conditional = false;
	static constexpr bool transpose = true;
	static constexpr bool scalar = true;

	template<typename param> static auto getScalar(const param& p)  { return p.left; }
	template<typename param> static auto getArray(const param& p)  { return p.right.array; }

};
//is a binary_expression (double already evaluated)

//transpose with inside scalar
template<class T, class lv, class rv, class U>
struct evaluate< unary_expression_transpose<U, binary_expression_scalar_R<T, mul, lv*, rv*>>> {

	static constexpr bool conditional = false;
	static constexpr bool transpose = true;
	static constexpr bool scalar = true;

	template<typename param> static auto getScalar(const param& p)  { return p.array.right; }
	template<typename param> static auto getArray(const param& p)  { return p.array.left; }
}; //is a binary_expression (expression by scalar)

template<class T, class lv, class rv, class U>
struct evaluate< unary_expression_transpose<U, binary_expression_scalar_L<T, mul, lv*, rv*>>> {

	static constexpr bool conditional = false;
	static constexpr bool transpose = true;
	static constexpr bool scalar = true;


	template<typename param> static auto getScalar(const param& p)  { return p.array.left; }
	template<typename param> static auto getArray(const param& p)  { return p.array.left; }
}; //is a binary_expression (scalar by expression)

}

#endif /* EXPRESSION_BINARY_DOTPRODUCT_IMPL2_H_ */
#endif
