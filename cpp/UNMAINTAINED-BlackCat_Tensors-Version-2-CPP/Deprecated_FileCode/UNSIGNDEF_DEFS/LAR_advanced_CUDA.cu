
#include <hip/hip_runtime.h>
//#include "BLACKCAT_GPU_MATHEMATICS.cuh"
//
//
//
//__global__ void  GPU_MATHEMATICS::correlation(unsigned* s, const unsigned* filter, const unsigned* signal, unsigned sz) {
//    	unsigned tmp = 0;
//    	for (unsigned i = 0; i < sz; ++i) {
//    		tmp += filter[i] * signal[i];
//    	}
//    	*s = tmp;
//    }
//
//
//__global__ void GPU_MATHEMATICS::correlation(unsigned* s, unsigned order, const unsigned* ranks,const unsigned* filter, const unsigned* f_ld,
//																		      const unsigned* signal, const unsigned* s_ld) {
//	//WORKS
////	if (order == 1) {
////    	unsigned tmp = 0;
////		correlation<<<256,256>>>(&tmp, filter, signal, ranks[0]);
////		*s += tmp;
////	} else {
////
////		for (unsigned i = 0; i < ranks[order-1]; ++i) {
////			correlation<<<256,256>>>(s, order-1, ranks, &filter[i * f_ld[order-1]], f_ld, &signal[i * s_ld[order-1]], s_ld);
////		}
////	}
//}
//
//
//
//
//__global__ void GPU_MATHEMATICS::cross_correlation(unsigned* s, unsigned cor_mv, const  unsigned* store_ld,
//												const unsigned* filter,const  unsigned * f_ld,const  unsigned* f_ranks, unsigned f_order,
//												const unsigned* signal, const unsigned * s_ld, const unsigned* s_ranks, unsigned s_order) {
//	if (cor_mv == 0) {
//		correlation<<<256,256>>>(s, f_order, f_ranks, filter, f_ld, signal, s_ld);
//	}
//
//	else if (s_order > f_order) {
//
//		unsigned s_id = 0;
//		for (unsigned i = 0; i < s_ranks[s_order-1]; ++i) {
//					cross_correlation<<<256,256>>>(s, cor_mv, store_ld, filter, f_ld, f_ranks, f_order, &signal[s_id], s_ld, s_ranks, s_order-1);
//
//					s_id += s_ld[s_order-1];
//				}
//	}
//
//
//	else {
//
//		unsigned store_id = 0;
//		unsigned s_id = 0;
//
//		unsigned mv_positions = s_ranks[cor_mv-1] + 1;
//		mv_positions -= f_order < cor_mv ? 1 : f_ranks[cor_mv-1];
//
//		for (unsigned i = 0; i < mv_positions; ++i) {
//			cross_correlation<<<256,256>>>(&s[store_id], cor_mv-1, store_ld, filter, f_ld, f_ranks, f_order, &signal[s_id], s_ld, s_ranks, s_order);
//
//			store_id += store_ld[cor_mv-1];
//			s_id += s_ld[cor_mv-1];
//		}
//	}
//}
//
//
//
//__global__ void GPU_MATHEMATICS::axpy(unsigned* store, const unsigned* store_ld, const unsigned* signal, const unsigned* signal_ld, const unsigned* signal_ranks, unsigned signal_order, unsigned scalar) {
//
//
//	if (signal_order == 1) {
//		for (unsigned i = 0; i < signal_ranks[0]; ++i) {
//			store[i] += signal[i] * scalar;
//
//		}
//	} else {
//
//
//		unsigned store_id = 0;
//		unsigned sig_id = 0;
//		for (unsigned i = 0; i < signal_ranks[signal_order-1]; ++i) {
//			axpy<<<256,256>>>(&store[store_id], store_ld, &signal[sig_id], signal_ld, signal_ranks, signal_order-1, scalar);
//			store_id += store_ld[signal_order-1];
//			sig_id += signal_ld[signal_order-1];
//
//		}
//	}
//}
//
//
//
//__global__ void GPU_MATHEMATICS::cc_filter_error(unsigned move_dimensions, unsigned* store, const unsigned* store_ld, const unsigned* store_ranks, unsigned store_order,
//										 	  const unsigned* error, const unsigned* error_ld, const unsigned* error_ranks, unsigned error_order,
//										 	  const unsigned* signal,const unsigned* signal_ld, const unsigned* signal_ranks, unsigned signal_order) {
//
//	if (move_dimensions > 0) {
//
//		unsigned move_positions = signal_ranks[move_dimensions-1] - store_ranks[move_dimensions-1] + 1;
//		unsigned sig_id = 0;
//		unsigned err_id = 0;
//
//		unsigned signal_base_index = 1;
//		unsigned error_base_index = 1;
//
//		for (int i = move_dimensions-1; i > 0; --i) {
//			error_base_index *= error_ld[i];
//			signal_base_index *= signal_ld[i];
//		}
//
//
//		for (unsigned i = 0; i < move_positions-1; ++i)  {
//			cc_filter_error<<<256,256>>>(move_dimensions-1, store, store_ld, store_ranks, store_order, &error[err_id], error_ld, error_ranks, error_order, &signal[sig_id], signal_ld, signal_ranks, signal_order);
//			sig_id += signal_base_index;
//			err_id += error_base_index;
//		}
//	} else {
//		axpy<<<256,256>>>(store, store_ld, signal, signal_ld, store_ranks, signal_order, *error);
//	}
//}
//
//
//
//__global__ void GPU_MATHEMATICS::cc_signal_error(unsigned move_dimensions, unsigned* store, const unsigned* store_ld, const unsigned* store_ranks, unsigned store_order,
//										 	  const unsigned* error, const unsigned* error_ld, const unsigned* error_ranks, unsigned error_order,
//										 	  const unsigned* filter,const unsigned* filter_ld, const unsigned* filter_ranks, unsigned filter_order) {
//
//	if (move_dimensions > 0) {
//
//		unsigned move_positions = move_dimensions-1 < error_order ?  error_ranks[move_dimensions-1] : 1;
//		unsigned err_id = 0;
//		unsigned sto_id = 0;
//
//
//		for (unsigned i = 0; i < move_positions; ++i)  {
//			cc_signal_error<<<256,256>>>(move_dimensions-1, &store[sto_id], store_ld, store_ranks, store_order, &error[err_id], error_ld, error_ranks, error_order, filter, filter_ld, filter_ranks, filter_order);
//			err_id += error_ld[move_dimensions-1];
//			sto_id += store_ld[move_dimensions-1];
//		}
//	} else {
//		axpy<<<256,256>>>(store, store_ld, filter, filter_ld, filter_ranks, filter_order, *error);
//	}
//}
////
////
////__global__ void GPU_MATHEMATICS::cc_error(unsigned move_dimensions, unsigned* store, const unsigned* store_ld, const unsigned* store_ranks, unsigned store_order,
////										 	  const unsigned* error, const unsigned* error_ld, const unsigned* error_ranks, unsigned error_order,
////										 	  const unsigned* signal,const unsigned* signal_ld, const unsigned* signal_ranks, unsigned signal_order) {
////
////	if (move_dimensions == 0) {
//////		std::cout << "calling axpy sub_kernel" << std::endl;
////		axpy(store, store_ld, signal, signal_ld, error_ranks, signal_order, error[0]);
////
////
////	} else if (signal_order > store_order) {
////		--signal_order;
////	//	std::cout << " moving loop ---------------------------------------" << std::endl;
////		unsigned sig_id = 0;
////		for (unsigned i = 0; i < signal_ranks[signal_order]; ++i) {
////			cc_error(move_dimensions, store, store_ld, store_ranks, store_order,
////						error, error_ld, error_ranks, error_order, &signal[sig_id], signal_ld, signal_ranks, signal_order);
////			sig_id += signal_ld[signal_order];
////		}
////	} else {
////		--move_dimensions;
////
////		unsigned mv_positions = signal_ranks[move_dimensions] + 1 - (store_order < move_dimensions ? 1 : store_ranks[move_dimensions]);
////		//mv_positions = 1;
//////		std::cout << " the number of mv positions = " << mv_positions << std::endl;
//////		std::cout << " move dim = " << move_dimensions << std::endl;
////		unsigned sig_id = 0;
////		unsigned store_id = 0;
////		unsigned error_id = 0;
////
////		for (unsigned i = 0; i < mv_positions; ++i) {
////			cc_error(move_dimensions, &store[store_id], store_ld, store_ranks, store_order,
////						&error[error_id], error_ld, error_ranks, error_order, &signal[sig_id], signal_ld, signal_ranks, signal_order);
////			sig_id += signal_ld[move_dimensions];
////			store_id += store_ld[move_dimensions];
////			error_id += error_ld[move_dimensions];
////		}
////	}
////
////}
