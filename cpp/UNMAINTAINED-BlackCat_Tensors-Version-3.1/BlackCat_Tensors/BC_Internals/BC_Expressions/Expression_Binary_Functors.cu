/*
 * BC_Expression_Binary_Functors.h
 *
 *  Created on: Dec 1, 2017
 *      Author: joseph
 */
#ifdef  __HIPCC__
#ifndef EXPRESSION_BINARY_FUNCTORS_H_
#define EXPRESSION_BINARY_FUNCTORS_H_
#include "../BlackCat_Internal_Definitions.h" //__BC_gcpu__

#include <iostream>
namespace BC {
		struct assign {

		template<class lv, class rv> __BC_gcpu__
		inline __attribute__((always_inline)) auto& operator ()(lv& l, rv r) const {
			return (l = r);
		}
	};

	struct combine {

		template<class lv, class rv> __BC_gcpu__
		inline __attribute__((always_inline)) auto operator ()(lv l, rv r) const {
			return l;
		}
	};


	struct add {

		template<class lv, class rv> __BC_gcpu__
		inline __attribute__((always_inline)) auto operator ()(lv l, rv r) const {
			return l + r;
		}
	};

	struct mul {
		template<class lv, class rv> __BC_gcpu__
		inline __attribute__((always_inline)) auto operator ()(lv l, rv r) const {
			return l * r;
		}
	};

	struct sub {
		template<class lv, class rv> __BC_gcpu__
		inline __attribute__((always_inline)) auto operator ()(lv l, rv r) const {
			return l - r;
		}
	};

	struct div {
		template<class lv, class rv> __BC_gcpu__
		inline __attribute__((always_inline)) auto operator ()(lv l, rv r) const {
			return l / r;
		}
	};
}


#endif /* EXPRESSION_BINARY_FUNCTORS_H_ */
#endif
