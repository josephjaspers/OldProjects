#include "hip/hip_runtime.h"
/*
 * Transpose_Wrapper.h
 *
 *  Created on: Dec 19, 2017
 *      Author: joseph
 */
#ifdef  __HIPCC__
#ifndef EXPRESSION_UNARY_MATRIXTRANSPOSITION_H_
#define EXPRESSION_UNARY_MATRIXTRANSPOSITION_H_
#include <iostream>
#include <cmath>
#include "Expression_Base.cu"
#include "../BC_MetaTemplateFunctions/Adhoc.h"
#include "../BlackCat_Internal_Definitions.h"

namespace BC {
template<class T, class functor_type>
struct unary_expression_transpose : expression<T, unary_expression_transpose<T, functor_type>>
{

	functor_type array;
	const bool vector = false;
	const int rows;
	const int cols;
	const int LD;

	unary_expression_transpose(functor_type p, int rows, int cols, int ld)
	: array(p), rows(rows), cols(cols), LD(ld), vector(cols == 1 || rows == 1) {}



private:
	template<class U> __BC_gcpu__
	static int bc_floor(U number) {
		return ((int)number) > number ? (int)(number - .5) : number; //if casting the number to an integer is greater (if it rounded up) subtract .5 and the round again
	}

public:


	__attribute__((always_inline)) __BC_gcpu__  auto operator [](int index) -> decltype(array[index]) {


		if (vector)
			return array[index];
		else
			return array[(int)(index / cols) + (index % cols) * LD];
	}
	__attribute__((always_inline))  __BC_gcpu__ auto operator[](int index) const  -> const decltype(array[index])  {

		if (vector)
			return array[index];
		else
			return array[(int)(index / cols) + (index % cols) * LD];
	}
};
}
#endif /* EXPRESSION_UNARY_MATRIXTRANSPOSITION_H_ */
#endif
