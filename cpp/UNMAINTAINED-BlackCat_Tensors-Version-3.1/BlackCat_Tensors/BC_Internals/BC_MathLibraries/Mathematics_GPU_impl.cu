#ifdef __HIPCC__
#ifndef BC_GPU_IMPL
#define BC_GPU_IMPL

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <limits>
#include <cstddef>
#include <type_traits>

namespace BC {
namespace gpu_impl {

template<typename T, typename J> __global__
static void fill(T& t, const J j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = j;
	}
}
template<typename T, typename J> __global__
static void fill(T& t, const J* j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = j[0];
	}
}

template<class T, class U, class V> __global__
static void scalarMul(T* t, U* u, V* v) {
	*t = u[0] * v[0];
}

template<typename T> __global__
static void zero(T& t, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = 0;
	}
}

template<class T, class J> __global__
static void copy(T t, const J j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = j[i];
	}
}

template<class T, class J> __global__
static void copyStructPtr(T t, J* j, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i] = (*j)[i];
	}
}

template<class T> __global__
static void eval(T t, int sz) {
	for (int i = 0; i < sz; ++i) {
		t[i];
	}
}

template<class T>
struct  _max {
	static constexpr T value = std::numeric_limits<T>::max();
};

template<typename T, typename J> __global__
static void randomize(T& t, J lower_bound, J upper_bound, int sz, int seed) {

	 hiprandState_t state;
	  hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
	              seed, /* the sequence number is only important with multiple cores */
	              1, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
	              &state);


	for (int i = 0; i < sz; ++i) {
		t[i] = hiprand(&state);
		t[i] /= 10000000000; //curand max value
		t[i] *= (upper_bound - lower_bound);
		t[i] += lower_bound;
	}
}

}


}

#endif
#endif //cudacc
