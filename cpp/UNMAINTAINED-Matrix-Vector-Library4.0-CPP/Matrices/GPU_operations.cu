#include "hip/hip_runtime.h"
#include "GPU_operations.cuh"


__global__ void Matrices_gpu::exp(double * sz, double * m1, double * m2, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = pow(m1[i], m2[i]);
	}
}
__global__ void Matrices_gpu::multiply(double * sz, double * m1, double * m2, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m1[i] * m2[i];
	}
}
__global__ void Matrices_gpu::divide(double * sz, double * m1, double * m2, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m1[i] / m2[i];
	}
}
__global__ void Matrices_gpu::add(double * sz, double * m1, double * m2, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m1[i] + m2[i];
	}
}
__global__ void Matrices_gpu::subtract(double * sz, double * m1, double * m2, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m1[i] - m2[i];
	}
}
__global__ void Matrices_gpu::exp(double * sz, double * m, double v, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = pow(m[i], v);
	}
}
__global__ void Matrices_gpu::multiply(double * sz, double * m, double v, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m[i] * v;
	}
}
__global__ void Matrices_gpu::divide(double * sz, double * m, double v, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m[i] / v;
	}
}
__global__ void Matrices_gpu::add(double * sz, double * m, double v, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m[i] + v;
	}
}
__global__ void Matrices_gpu::subtract(double * sz, double * m, double v, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m[i] - v;
	}
}
__global__ void Matrices_gpu::dot(double * sz, double * m1, int mat1_length, int mat1_width, double * m2, int mat2_length, int mat2_width)
{
	int save_size = mat1_width * mat2_length;
	int stride_size = mat1_length;
	int n = save_size * stride_size;

	for (int i = 0; i < n; ++i)
		//for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
	{
		int save_index = floor((double)i / (mat1_length));
		int m1_index = i % mat1_length + floor((double)save_index / mat1_width) * mat1_length;
		int m2_index = (i % mat2_width) * mat2_length + (save_index % mat2_length);
		sz[save_index] += m1[m1_index] * m2[m2_index];
	}
}
__global__ void Matrices_gpu::dot_transpose(double * s, double * m1, int mat1_length, int mat1_width, double * m2, int mat2_length, int mat2_width)
{
	int save_size = mat1_length * mat2_length;
	int stride_size = mat2_width;
	int n = save_size * stride_size;

	for (int i = 0; i < n; ++i)
	{
		int save_index = floor((double)i / (mat1_width));
		int m1_index = (i % mat1_width) * mat1_length + ((int)floor((double)(i / mat1_length) / mat2_width));
		int m2_index = (i % mat2_width) * mat2_length + (save_index % mat2_length);

		s[save_index] += m1[m1_index] * m2[m2_index];
	}
}
__global__ void Matrices_gpu::dot_Vector_Vector(double * sz, double * v1, double * v2, int n)
{
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			sz[i] += v1[i] * v2[j];
		}
	}
}
__global__ void Matrices_gpu::dot_Vector_Vector_inner(double * s, double * v1, double * v2, int n)
{
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		*s += v1[i] * v2[i];
	}
}

__global__ void Matrices_gpu::dot_Matrix_T_Matrix(double * sz, double * m1, int mat1_length, int mat1_width, double * m2, int mat2_length, int mat2_width)
{
	int save_size = mat1_width * mat2_length;
	int stride_size = mat1_length;
	int n = save_size * stride_size;

	for (int i = 0; i < n; ++i)
		//for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
	{
		int save_index = floor((double)i / (mat1_width));
		int m1_index = (i % mat1_width) * mat1_length + (save_index % mat1_length);
		int m2_index = (i % mat2_width) * mat2_length + (save_index % mat2_length);
		sz[save_index] += m1[m1_index] * m2[m2_index];
	}
}

__global__ void Matrices_gpu::dot_Matrix_Matrix_T(double * sz, double * m1, int mat1_length, int mat1_width, double * m2, int mat2_length, int mat2_width)
{
	int save_size = mat1_width * mat2_length;
	int stride_size = mat1_length;
	int n = save_size * stride_size;

	for (int i = 0; i < n; ++i)
		//for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
	{
		int save_index = floor((double)i / (mat1_length));
		int m1_index = i % mat1_length + floor((double)save_index / mat1_width) * mat1_length;
		int m2_index = i % mat2_length + floor((double)save_index / mat2_width) * mat2_length;
		sz[save_index] += m1[m1_index] * m2[m2_index];
	}
}

__global__ void Matrices_gpu::dot_Matrix_T_Matrix_T(double * sz, double * m1, int mat1_length, int mat1_width, double * m2, int mat2_length, int mat2_width)
{
	int save_size = mat1_width * mat2_length;
	int stride_size = mat1_length;
	int n = save_size * stride_size;

	for (int i = 0; i < n; ++i)
		//for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
	{
		int save_index = floor((double)i / (mat1_width));
		int m1_index = (i % mat1_width) * mat1_length + (save_index % mat1_length);
		int m2_index = i % mat2_length + floor((double)save_index / mat2_width) * mat2_length;
		sz[save_index] += m1[m1_index] * m2[m2_index];
	}//
}

__global__ void equal(int& boolean, double* m1, double* m2, int n) {
	//Grid stride
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		if (m1[i] != m2[i]) {
			boolean = 0;
			return;
		}
	}
}
__global__ void Matrices_gpu::copy(double* sz, double* m1, int n) {
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		sz[i] = m1[i];
	}
}
__global__ void Matrices_gpu::equal(bool * boolean, double * m1, double * m2, int n)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		if (m1[i] != m2[i]) {
			boolean = false;
			return;
		}
	}
}
__global__ void Matrices_gpu::fill(double * m1, double value, int n)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < n;
		i += blockDim.x * gridDim.x)
	{
		m1[i] = value;
	}
}

__global__ void Matrices_gpu::transpose(double * s, double * m, int x, int y)
{
	for (int i = 0; i < x * y; ++i) {
		s[i] = m[i % y * x + (int)floor((double)i / y) % x];
	}
}

__global__ void Matrices_gpu::reshape(double * s, double * m, int x, int y)
{
	for (int i = blockIdx.x * blockDim.x + threadIdx.x;
		i < (x * y);
		i += blockDim.x * gridDim.x)
	{
		for (int i = 0; i < x * y; ++i) {
			s[i] = m[i % y * x + (int)floor((double)i / y) % x];
		}
	}
}
